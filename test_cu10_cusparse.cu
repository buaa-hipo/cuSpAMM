#include "common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hipblas.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <omp.h>
#include <assert.h>
#include "tool.cu"

using namespace std;
/*
 * This is an example demonstrating usage of the cuSPARSE library to perform a
 * sparse matrix-vector multiplication on randomly generated data.
*/

/*
 * M = # of rows
 * N = # of columns
 */

void print_partial_matrix(float *M, int nrows, int ncols, int max_row,
         int max_col)
 {
     int row, col;
 
     for (row = 0; row < max_row; row++)
     {
         for (col = 0; col < max_col; col++)
         {
             printf("%2.2f ", M[row * ncols + col]);
         }
         printf("...\n");
     }
     printf("...\n");
 }
 
 
double test_cusparse_real(int M, int N, mytype thresh, mytype* origin, mytype* dOri)
{
     mytype *A, *dA, *dB;
     mytype *C, *dC;
     int *dANnzPerRow;
     mytype *dCsrValA;
     int *dCsrRowPtrA;
     int *dCsrColIndA;
     int *dBNnzPerRow;
     mytype *dCsrValB;
     int *dCsrRowPtrB;
     int *dCsrColIndB;
     int *dCNnzPerRow;
     mytype *dCsrValC;
     int *dCsrRowPtrC;
     int *dCsrColIndC;
     int totalANnz,totalBNnz;
     mytype alpha = 3.0f;
     mytype beta = 4.0f;
     hipsparseHandle_t handle = 0;
     hipsparseMatDescr_t Adescr = 0;
     hipsparseMatDescr_t Bdescr = 0;
     hipsparseMatDescr_t Cdescr = 0;
     hipEvent_t start, stop;
    float elapsed = 0.0;
    double sum=0.0;
    hipsparseStatus_t stat = HIPSPARSE_STATUS_SUCCESS;
 
    // Generate input
    A = (mytype *)malloc(sizeof(mytype) * M * M);
    fill(A, A + M * M, 0.0);
    int trueANnz = getMatrixFromMTX(A, M, M, "../matrix/decay1024_3.mtx", thresh);
    cout << "ratio of non-zero: " << trueANnz << " / " << M * M << " = " << (double)(trueANnz)/(double)(M * M) << endl;
    C = (mytype *)malloc(sizeof(mytype) * M * M);
 
    // Create the cuSPARSE handle
    hipsparseCreate(&handle);
 
    // Allocate device memory for vectors and the dense form of the matrix A
    hipMalloc((void **)&dA, sizeof(mytype) * M * N);
    hipMalloc((void **)&dB, sizeof(mytype) * M * N);
    hipMalloc((void **)&dC, sizeof(mytype) * M * M);
    hipMalloc((void **)&dANnzPerRow, sizeof(int) * M);
    hipMalloc((void **)&dBNnzPerRow, sizeof(int) * M);
    assert(hipGetLastError() == hipSuccess);
 
    // Construct a descriptor of the matrix A
    hipsparseCreateMatDescr(&Adescr);
    hipsparseSetMatType(Adescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(Adescr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseCreateMatDescr(&Bdescr);
    hipsparseSetMatType(Bdescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(Bdescr, HIPSPARSE_INDEX_BASE_ZERO);

    hipsparseCreateMatDescr(&Cdescr);
    hipsparseSetMatType(Cdescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
    hipsparseSetMatIndexBase(Cdescr, HIPSPARSE_INDEX_BASE_ZERO);
 
    // Transfer the input vectors and dense matrix A to the device
    hipMemcpy(dA, A, sizeof(mytype) * M * M, hipMemcpyHostToDevice);
    hipMemcpy(dB, A, sizeof(mytype) * M * M, hipMemcpyHostToDevice);
    hipMemset(dC, 0x00, sizeof(mytype) * M * M);
    assert(hipGetLastError() == hipSuccess);
 
    // Compute the number of non-zero elements in A
    hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, M, M, Adescr,
                                dA, M, dANnzPerRow, &totalANnz);
    hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, M, M, Bdescr,
                                    dB, M, dBNnzPerRow, &totalBNnz);
 
    if (totalANnz != trueANnz)
    {
        fprintf(stderr, "Difference detected between cuSPARSE NNZ and true "
                "value: expected %d but got %d\n", trueANnz, totalANnz);
        return 1;
    }
 
    // Allocate device memory to store the sparse CSR representation of A
    hipMalloc((void **)&dCsrValA, sizeof(mytype) * totalANnz);
    hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (M + 1));
    hipMalloc((void **)&dCsrColIndA, sizeof(int) * totalANnz);
    hipMalloc((void **)&dCsrValB, sizeof(mytype) * totalBNnz);
    hipMalloc((void **)&dCsrRowPtrB, sizeof(int) * (M + 1));
    hipMalloc((void **)&dCsrColIndB, sizeof(int) * totalBNnz);
    assert(hipGetLastError() == hipSuccess);

    stat = hipsparseSdense2csr(handle, M, M, Adescr, dA, M, dANnzPerRow,
        dCsrValA, dCsrRowPtrA, dCsrColIndA);
    stat = hipsparseSdense2csr(handle, M, M, Bdescr, dB, M, dBNnzPerRow,
        dCsrValB, dCsrRowPtrB, dCsrColIndB);
    assert(stat == HIPSPARSE_STATUS_SUCCESS);

     int nnzC;
     int* nnzTotalDevHostPtr = &nnzC;
     hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
     hipMalloc((void**)&dCsrRowPtrC, sizeof(int) * (M + 1));

     hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	hipsparseOperation_t transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;

	 hipsparseXcsrgemmNnz(handle, transA, transB, M, M, M,
		Adescr, trueANnz, dCsrRowPtrA, dCsrColIndA,
		Bdescr, trueANnz, dCsrRowPtrB, dCsrColIndB,
		Cdescr, dCsrRowPtrC, nnzTotalDevHostPtr);
    
    if (NULL != nnzTotalDevHostPtr) {
        nnzC = *nnzTotalDevHostPtr;
            // printf("hahaha\n");
    }
    printf("nnzC = %d\n",nnzC);

    hipMalloc((void **)&dCsrValC, sizeof(mytype) * nnzC);
    hipMalloc((void **)&dCsrRowPtrC, sizeof(int) * (M + 1));
    hipMalloc((void **)&dCsrColIndC, sizeof(int) * nnzC);
    assert(hipGetLastError() == hipSuccess);
 
    
    // printf("\n***计时***\n");
    
    printf("start gemm\n");

    for(int i=0; i < WARMUP+TESTTIME; i++) {
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        stat = hipsparseScsrgemm(handle, transA, transB, M, M, M,
            Adescr, totalANnz,
            dCsrValA, dCsrRowPtrA, dCsrColIndA,
            Bdescr, totalBNnz,
            dCsrValB, dCsrRowPtrB, dCsrColIndB,
            Cdescr,
            dCsrValC, dCsrRowPtrC, dCsrColIndC);
        assert(stat == HIPSPARSE_STATUS_SUCCESS);

        hipDeviceSynchronize();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        elapsed /= 1000.0f;
        if(i>2) sum += elapsed; 
    }
    double cusparsetime=sum/((TESTTIME));
    // printf("cuSPARSE N=%d nnzA=%d(nnz rate=%f) nnzC=%d(nnz rate=%f) \n平均执行时间=%fs\n",M,totalANnz,(double)totalANnz/M/M,nnzC,(double)nnzC/M/M,cusparsetime);
    printf("cuSPARSE M=%d nnzA=%d(nnz rate=%f) \n平均执行时间=%fs\n",M,totalANnz,(double)totalANnz/M/M,cusparsetime);

    stat = hipsparseScsr2dense(handle, M, M, Cdescr, dCsrValC, dCsrRowPtrC, dCsrColIndC, dC, M);
    assert(stat == HIPSPARSE_STATUS_SUCCESS);

    // Copy the result vector back to the host
    hipMemcpy(C, dC, sizeof(mytype) * M * M, hipMemcpyDeviceToHost);
    assert(hipGetLastError() == hipSuccess);
 
    printf("C:\n");
    print_partial_matrix(C, M, M, 10, 10);

    // -------------------------------------------------------------------------
    printf("perform gemm in dense format\n");
    // perform matmul in dense format
    mytype* hC2 = (mytype *)malloc(sizeof(mytype) * M * M);
    mytype* dC2;

    hipMalloc((void**) &dC2, M * M * sizeof(mytype));

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    #if !USINGHALF
    // cublasXtSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, m, n, k, alpha, AA, k, BB, n, beta, C, m);
    CHECK_CUBLAS( hipblasSgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, M, M, &alpha, dOri, M, dOri, M, &beta, dC2, M) ) 
    #else
    CHECK_CUBLAS( hipblasHgemm(cublas_handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, M, M, &alpha, dOri, M, dOri, M, &beta, dC2, M) )
    #endif

    hipMemcpy(hC2, dC2, M * M * sizeof(mytype), hipMemcpyDeviceToHost);
    assert(hipGetLastError() == hipSuccess);

    // --------------------------------------------------------------------------
    // check hC with hC2
    COUNTERRTRANS(hC2, C, M, M)


 
     free(A);
     free(C);
 
     hipFree(dA);
     hipFree(dC);
     hipFree(dANnzPerRow);
     hipFree(dCsrValA);
     hipFree(dCsrRowPtrA);
     hipFree(dCsrColIndA);
    assert(hipGetLastError() == hipSuccess);
 
     hipsparseDestroyMatDescr(Adescr);
     hipsparseDestroy(handle);
 
     return cusparsetime;
}

int main(int argc, char **argv) {
    // vector<int> sz_vec{1024, 1024, 1024, 8192, 8192, 8192};
    // vector<mytype> tr_vec{0.036, 0.038, 0.04, 0.031, 0.033, 0.039};
    vector<mytype> tr_vec{1e-10, 1e-8, 1e-6, 1e-4, 1e-2};

    int M = 1024;
    mytype* origin = (mytype*) malloc(sizeof(mytype) * M * M);
    getMatrixFromMTX(origin, M, M, "../matrix/decay1024_3.mtx", 0);
    mytype* dOri;
    CHECK_CUDA( hipMalloc((void**) &dOri, M * M * sizeof(mytype))   )
    hipMemcpy(dOri, origin, sizeof(mytype) * M * M, hipMemcpyHostToDevice);


    ofstream out("cusparse_real.csv");
    for (int i = 1; i < 4; i ++) {

        cout << (mytype)tr_vec[i] << " start " << endl;

        double cost = test_cusparse_real(M, M, tr_vec[i],origin, dOri);
        cout << (mytype)tr_vec[i] << " cost_time: " << cost << endl << endl;
        out << (mytype)tr_vec[i] << ", " << cost << endl;
    }
    return 0;
}
