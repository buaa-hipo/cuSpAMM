#include "common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <omp.h>
#include <assert.h>

using namespace std;

// #include "main.h"
/*
 * This is an example demonstrating usage of the cuSPARSE library to perform a
 * sparse matrix-vector multiplication on randomly generated data.
*/

/*
 * M = # of rows
 * N = # of columns
 */

 
double test_cusparse_csr(int M, int N, mytype thresh)
{
    mytype *hA, *dA;

    mytype alpha = 3.0f;
    mytype beta = 4.0f;
    hipsparseHandle_t handle = 0;
    hipsparseDnMatDescr_t Adescr_dn;
    hipsparseSpMatDescr_t Adescr, Bdescr;
    hipsparseSpMatDescr_t Cdescr = 0;

    hipDataType        computeType = cudatatype;

    // Create the cuSPARSE handle
    CHECK_CUSPARSE(hipsparseCreate(&handle));
 
//-------------------------------------------------------------------
    // prepare A
    // Generate input
    // srand(9384);
    hA = (mytype *)malloc(sizeof(mytype) * M * M);
    int A_nnz = getDecayMatrixAlg(hA, 0.1, 0.1, M, M, thresh);
    cout << "ratio of non-zero: " << A_nnz << " / " << M * M << " = " << (double)(A_nnz)/(double)(M * M) << endl;
 
    // Allocate device memory for the dense form of the matrix A
    hipMalloc((void **)&dA, sizeof(mytype) * M * M);
 
    // Transfer the input dense matrix A to the device
    hipMemcpy(dA, hA, sizeof(mytype) * M * M, hipMemcpyHostToDevice);


    // Create dense matrix A
    CHECK_CUSPARSE( hipsparseCreateDnMat(&Adescr_dn, M, M, M, dA,
                                        cudatatype, HIPSPARSE_ORDER_ROW) )
    int   *dA_csr_offsets, *dA_csr_columns;
    mytype *dA_csr_values;

    // Allocate device memory to store the sparse CSR representation of A
    CHECK_CUDA( hipMalloc((void**) &dA_csr_offsets, (M + 1) * sizeof(int)) )
    // Create sparse matrix A in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&Adescr, M, M, 0,
                                      dA_csr_offsets, NULL, NULL, 
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, cudatatype) )

//----------------------------------------------------------------------------
    // dense to csr (cuda v11.2.0 or above required)
    size_t bufferSize0 = 0;
    void* dBuffer0 = NULL;
    CHECK_CUSPARSE (
        hipsparseDenseToSparse_bufferSize(handle, 
                                        Adescr_dn, 
                                        Adescr, 
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        &bufferSize0)
    )
    CHECK_CUDA( hipMalloc(&dBuffer0, bufferSize0) )

    CHECK_CUSPARSE (
        hipsparseDenseToSparse_analysis(handle, 
                                        Adescr_dn, 
                                        Adescr, 
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer0)
    )

    // get number of non-zero elements
    int64_t num_rows_tmp, num_cols_tmp, nnz;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(Adescr, &num_rows_tmp, &num_cols_tmp,
                                         &nnz) )

    // allocate CSR column indices and values
    CHECK_CUDA( hipMalloc((void**) &dA_csr_columns, nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dA_csr_values,  nnz * sizeof(mytype)) )

    // reset offsets, column indices, and values pointers
    CHECK_CUSPARSE( hipsparseCsrSetPointers(Adescr, dA_csr_offsets, dA_csr_columns,
                                           dA_csr_values) )
    // excute dense to sparse conversion
    CHECK_CUSPARSE (
        hipsparseDenseToSparse_convert(handle, 
                                        Adescr_dn, 
                                        Adescr, 
                                        HIPSPARSE_DENSETOSPARSE_ALG_DEFAULT,
                                        dBuffer0)
    )
    
    CHECK_CUSPARSE( hipsparseDestroyDnMat(Adescr_dn) )
    CHECK_CUDA( hipFree(dA) )

//----------------------------------------------------------------------------
    // prepare B
    int   *dB_csr_offsets0, *dB_csr_columns0;
    int   *dB_csr_offsets, *dB_csr_columns;
    mytype *dB_csr_values0, *dB_csr_values;


    // Allocate device memory to store the sparse CSR representation of B
    CHECK_CUDA( hipMalloc((void**) &dB_csr_offsets0, (M + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_csr_columns0, nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dB_csr_values0,  nnz * sizeof(mytype)) )

    CHECK_CUDA( hipMalloc((void**) &dB_csr_offsets, (M + 1) * sizeof(int)) )
    CHECK_CUDA( hipMalloc((void**) &dB_csr_columns, nnz * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dB_csr_values,  nnz * sizeof(mytype)) )


    CHECK_CUDA( hipMemcpy(dB_csr_offsets0, dA_csr_offsets, sizeof(int) * (M + 1), hipMemcpyDeviceToDevice) )
    CHECK_CUDA( hipMemcpy(dB_csr_columns0, dA_csr_columns, sizeof(int) * nnz, hipMemcpyDeviceToDevice) )
    CHECK_CUDA( hipMemcpy(dB_csr_values0, dA_csr_values, sizeof(mytype) * nnz, hipMemcpyDeviceToDevice) )

    size_t tp_buffer_size = 0;
    void* tp_buffer = NULL;
    // transpose B (csc format is the transposed B)
    CHECK_CUSPARSE( hipsparseCsr2cscEx2_bufferSize(handle, M, M, nnz, 
                                dB_csr_values0,     // nnz
                                dB_csr_offsets0,    // M + 1
                                dB_csr_columns0,    // nnz
                                dB_csr_values,      // nnz
                                dB_csr_offsets,     // M + 1
                                dB_csr_columns,     // nnz
                                cudatatype, 
                                HIPSPARSE_ACTION_NUMERIC, 
                                HIPSPARSE_INDEX_BASE_ZERO,
                                HIPSPARSE_CSR2CSC_ALG1, 
                                &tp_buffer_size)
    )

    CHECK_CUDA( hipMalloc((void**) &tp_buffer, tp_buffer_size * sizeof(mytype))   )

    CHECK_CUSPARSE( hipsparseCsr2cscEx2(handle, M, M, nnz, 
                                dB_csr_values0,     // nnz
                                dB_csr_offsets0,    // M + 1
                                dB_csr_columns0,    // nnz
                                dB_csr_values,      // nnz
                                dB_csr_offsets,     // M + 1
                                dB_csr_columns,     // nnz
                                cudatatype, 
                                HIPSPARSE_ACTION_NUMERIC, 
                                HIPSPARSE_INDEX_BASE_ZERO,
                                HIPSPARSE_CSR2CSC_ALG1, 
                                tp_buffer)
    )

    // Create sparse matrix B in CSR format
    CHECK_CUSPARSE( hipsparseCreateCsr(&Bdescr, M, M, nnz,
                                      dB_csr_offsets, dB_csr_columns, dB_csr_values, 
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, cudatatype) )

    CHECK_CUDA( hipFree(dB_csr_columns0) )
    CHECK_CUDA( hipFree(dB_csr_offsets0) )
    CHECK_CUDA( hipFree(dB_csr_values0) )

//----------------------------------------------------------------------------
    // prepare C
    // mytype* hC = (mytype *)malloc(sizeof(mytype) * M * M);

    int *dC_csrOffsets, *dC_columns;
    mytype *dC_values;
    // allocate C offsets
    CHECK_CUDA( hipMalloc((void**) &dC_csrOffsets,
                           (M + 1) * sizeof(int)) )
    CHECK_CUSPARSE( hipsparseCreateCsr(&Cdescr, M, M, 0,
                                      NULL, NULL, NULL,
                                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                                      HIPSPARSE_INDEX_BASE_ZERO, cudatatype) )


//------------------------------------------------------------------------------------------
    // SpGEMM Computation
    hipsparseSpGEMMDescr_t spgemmDesc;
    CHECK_CUSPARSE( hipsparseSpGEMM_createDescr(&spgemmDesc) )

    // Only opA, opB equal to HIPSPARSE_OPERATION_NON_TRANSPOSE are supported
    hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;

    void*  dBuffer1    = NULL, *dBuffer2   = NULL;
    size_t bufferSize1 = 0,    bufferSize2 = 0;
    // ask bufferSize1 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, Adescr, Bdescr, &beta, Cdescr,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer1, bufferSize1) )
    // inspect the matrices A and B to understand the memory requirement for
    // the next step
    CHECK_CUSPARSE(
        hipsparseSpGEMM_workEstimation(handle, opA, opB,
                                      &alpha, Adescr, Bdescr, &beta, Cdescr,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                      spgemmDesc, &bufferSize1, dBuffer1) )

    // ask bufferSize2 bytes for external memory
    CHECK_CUSPARSE(
        hipsparseSpGEMM_compute(handle, opA, opB,
                               &alpha, Adescr, Bdescr, &beta, Cdescr,
                               computeType, HIPSPARSE_SPGEMM_DEFAULT,
                               spgemmDesc, &bufferSize2, NULL) )
    CHECK_CUDA( hipMalloc((void**) &dBuffer2, bufferSize2) )

    // printf("\n***计时***\n");
    hipEvent_t start, stop;
    float elapsed = 0.0;
    double sum=0.0;

    for(int i=0; i < WARMUP+TESTTIME; i++){
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
        
        // compute the intermediate product of A * B
        CHECK_CUSPARSE( hipsparseSpGEMM_compute(handle, opA, opB,
                                            &alpha, Adescr, Bdescr, &beta, Cdescr,
                                            computeType, HIPSPARSE_SPGEMM_DEFAULT,
                                            spgemmDesc, &bufferSize2, dBuffer2) )


        hipDeviceSynchronize();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        elapsed /= 1000.0f;
        if(i>=WARMUP) sum += elapsed; 
    }
    double cusparsetime=sum/((TESTTIME));
    // printf("cuSPARSE N=%d nnzA=%d(nnz rate=%f) nnzC=%d(nnz rate=%f) \n平均执行时间=%fs\n",N,totalANnz,(double)totalANnz/M/M,nnzC,(double)nnzC/M/M,cusparsetime);
    // printf("cuSPARSE N=%d nnzA=%d(nnz rate=%f) \n平均执行时间=%fs\n",N,totalANnz,(double)totalANnz/M/M,cusparsetime);

    // get matrix C non-zero entries C_nnz1
    int64_t C_num_rows1, C_num_cols1, C_nnz1;
    CHECK_CUSPARSE( hipsparseSpMatGetSize(Cdescr, &C_num_rows1, &C_num_cols1,
                                         &C_nnz1) )
    // allocate matrix C
    CHECK_CUDA( hipMalloc((void**) &dC_columns, C_nnz1 * sizeof(int))   )
    CHECK_CUDA( hipMalloc((void**) &dC_values,  C_nnz1 * sizeof(float)) )
    // update matC with the new pointers
    CHECK_CUSPARSE(
        hipsparseCsrSetPointers(Cdescr, dC_csrOffsets, dC_columns, dC_values) )

    // if beta != 0, hipsparseSpGEMM_copy reuses/updates the values of dC_values

    // copy the final products to the matrix C
    CHECK_CUSPARSE(
        hipsparseSpGEMM_copy(handle, opA, opB,
                            &alpha, Adescr, Adescr, &beta, Cdescr,
                            computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc) )

    // destroy matrix/vector descriptors
    CHECK_CUSPARSE( hipsparseSpGEMM_destroyDescr(spgemmDesc) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(Adescr) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(Bdescr) )
    CHECK_CUSPARSE( hipsparseDestroySpMat(Cdescr) )
    CHECK_CUSPARSE( hipsparseDestroy(handle) )
 
    return cusparsetime;
}


int main(int argc, char **argv) {
    vector<int> sz_vec{1024, 1024, 1024, 8192, 8192, 8192};
    vector<mytype> tr_vec{0.036, 0.038, 0.04, 0.031, 0.033, 0.039};
    ofstream out("cusparse_csr_time.csv");
    for (int i = 0; i < 6; i ++) {
        double cost = test_cusparse_csr(sz_vec[i], sz_vec[i], tr_vec[i]);
        cout << sz_vec[i] << ", " << (float)tr_vec[i] << ": " << cost << endl;
        out << sz_vec[i] << ", " << (float)tr_vec[i] << ", " << cost << endl;
    }
    return 0;
}
