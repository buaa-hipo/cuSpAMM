#include "common.h"
#include <stdio.h>
#include <stdlib.h>
#include <hipsparse.h>
#include <hip/hip_runtime.h>
#include "main.h"
/*
 * This is an example demonstrating usage of the cuSPARSE library to perform a
 * sparse matrix-vector multiplication on randomly generated data.
*/

/*
 * M = # of rows
 * N = # of columns
 */
 
 //生成25%稠密矩阵
 int generate_random_dense_matrix(int M, int N, float **outA)
 {
     int i, j;
     double rMax = (double)RAND_MAX;
     float *A = (float *)malloc(sizeof(float) * M * N);
     int totalNnz = 0;
 
     for (j = 0; j < N; j++)
     {
         for (i = 0; i < M; i++)
         {
            
             float *curr = A + (j * M + i);
 
             if (i >= 0.25*M)
             {
                 *curr = 0.0f;
             }
             else
             {
                int r = rand( );
                 double dr = (double)r;
                 *curr = (dr / rMax) * 100.0;
             }
 
             if (*curr != 0.0f)
             {
                 totalNnz++;
             }
         }
     }
 
     *outA = A;
     return totalNnz;
 }
 
 void print_partial_matrix(float *M, int nrows, int ncols, int max_row,
         int max_col)
 {
     int row, col;
 
     for (row = 0; row < max_row; row++)
     {
         for (col = 0; col < max_col; col++)
         {
             printf("%2.2f ", M[row * ncols + col]);
         }
         printf("...\n");
     }
     printf("...\n");
 }
 
 int main(int argc, char **argv)
 {
     float *A, *dA;
     float *B, *dB;
     float *C, *dC;
     int *dANnzPerRow;
     float *dCsrValA;
     int *dCsrRowPtrA;
     int *dCsrColIndA;
     int *dCNnzPerRow;
     float *dCsrValC;
     int *dCsrRowPtrC;
     int *dCsrColIndC;
     int totalANnz;
     float alpha = 3.0f;
     float beta = 4.0f;
     hipsparseHandle_t handle = 0;
     hipsparseMatDescr_t Adescr = 0;
     hipsparseMatDescr_t Cdescr = 0;
     hipEvent_t start, stop;
    float elapsed = 0.0;
    double sum=0.0;
 
     // Generate input
     srand(9384);
     int trueANnz = generate_random_dense_matrix(M, N, &A);
     int trueBNnz = generate_random_dense_matrix(N, M, &B);
     C = (float *)malloc(sizeof(float) * M * M);
 
     // Create the cuSPARSE handle
     CHECK_CUSPARSE(hipsparseCreate(&handle));
 
     // Allocate device memory for vectors and the dense form of the matrix A
     hipMalloc((void **)&dA, sizeof(float) * M * N);
     hipMalloc((void **)&dB, sizeof(float) * N * M);
     hipMalloc((void **)&dC, sizeof(float) * M * M);
     hipMalloc((void **)&dANnzPerRow, sizeof(int) * M);
 
     // Construct a descriptor of the matrix A
     hipsparseCreateMatDescr(&Adescr);
     hipsparseSetMatType(Adescr, HIPSPARSE_MATRIX_TYPE_GENERAL);
     hipsparseSetMatIndexBase(Adescr, HIPSPARSE_INDEX_BASE_ZERO);

     CHECK_CUSPARSE(hipsparseCreateMatDescr(&Cdescr));
     CHECK_CUSPARSE(hipsparseSetMatType(Cdescr, HIPSPARSE_MATRIX_TYPE_GENERAL));
     CHECK_CUSPARSE(hipsparseSetMatIndexBase(Cdescr, HIPSPARSE_INDEX_BASE_ZERO));
 
     // Transfer the input vectors and dense matrix A to the device
     hipMemcpy(dA, A, sizeof(float) * M * N, hipMemcpyHostToDevice);
     hipMemcpy(dB, B, sizeof(float) * N * M, hipMemcpyHostToDevice);
     hipMemset(dC, 0x00, sizeof(float) * M * M);
 
     // Compute the number of non-zero elements in A
     CHECK_CUSPARSE(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_ROW, M, N, Adescr,
                                 dA, M, dANnzPerRow, &totalANnz));
 
     if (totalANnz != trueANnz)
     {
         fprintf(stderr, "Difference detected between cuSPARSE NNZ and true "
                 "value: expected %d but got %d\n", trueANnz, totalANnz);
         return 1;
     }
 
     // Allocate device memory to store the sparse CSR representation of A
     hipMalloc((void **)&dCsrValA, sizeof(float) * totalANnz);
     hipMalloc((void **)&dCsrRowPtrA, sizeof(int) * (M + 1));
     hipMalloc((void **)&dCsrColIndA, sizeof(int) * totalANnz);
     CHECK_CUSPARSE(hipsparseSdense2csr(handle, M, N, Adescr, dA, M, dANnzPerRow,
        dCsrValA, dCsrRowPtrA, dCsrColIndA));

     int nnzC;
     int* nnzTotalDevHostPtr = &nnzC;
     hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);
     hipMalloc((void**)&dCsrRowPtrC, sizeof(int) * (M + 1));

     hipsparseOperation_t transA = HIPSPARSE_OPERATION_TRANSPOSE;
	hipsparseOperation_t transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
	 hipsparseXcsrgemmNnz(handle, transA, transB, M, N, K,
		Adescr, trueANnz, dCsrRowPtrA, dCsrColIndA,
		Adescr, trueANnz, dCsrRowPtrA, dCsrColIndA,
		Cdescr, dCsrRowPtrC, nnzTotalDevHostPtr);
        if (NULL != nnzTotalDevHostPtr) {
            nnzC = *nnzTotalDevHostPtr;
            // printf("hahaha\n");
        }
        // printf("%d\n",nnzC);

        hipMalloc((void **)&dCsrValC, sizeof(float) * nnzC);
        hipMalloc((void **)&dCsrRowPtrC, sizeof(int) * (M + 1));
        hipMalloc((void **)&dCsrColIndC, sizeof(int) * nnzC);
 
     
 
    // // Convert A from a dense formatting to a CSR formatting, using the GPU
    // CHECK_CUSPARSE(hipsparseSdense2csr(handle, M, N, Adescr, dA, M, dANnzPerRow,
    //     dCsrValA, dCsrRowPtrA, dCsrColIndA));
    // printf("\n***计时***\n");
    
    for(int i=0;i<10;i++){
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        

    //     // CHECK_CUSPARSE(hipsparseScsrmm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, M,
    //     //     M, N, totalANnz, &alpha, Adescr, dCsrValA,
    //     //     dCsrRowPtrA, dCsrColIndA, dB, N, &beta, dC,
    //     //     // M));

        hipsparseScsrgemm(handle, transA, transB, M, N, K,
            Adescr, totalANnz,
            dCsrValA, dCsrRowPtrA, dCsrColIndA,
            Adescr, totalANnz,
            dCsrValA, dCsrRowPtrA, dCsrColIndA,
            Cdescr,
            dCsrValC, dCsrRowPtrC, dCsrColIndC);

        hipDeviceSynchronize();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        elapsed /= 1000.0f;
        if(i>2) sum += elapsed; 
    }
    double cusparsetime=sum/((7));
    // printf("cuSPARSE N=%d nnzA=%d(nnz rate=%f) nnzC=%d(nnz rate=%f) \n平均执行时间=%fs\n",N,totalANnz,(double)totalANnz/M/M,nnzC,(double)nnzC/M/M,cusparsetime);
    printf("cuSPARSE N=%d nnzA=%d(nnz rate=%f) \n平均执行时间=%fs\n",N,totalANnz,(double)totalANnz/M/M,cusparsetime);
 
    //  // Copy the result vector back to the host
    //  CHECK(hipMemcpy(C, dC, sizeof(float) * M * M, hipMemcpyDeviceToHost));
 
    // //  printf("C:\n");
    // //  print_partial_matrix(C, M, M, 10, 10);
 
    //  free(A);
    //  free(B);
    //  free(C);
 
    //  CHECK(hipFree(dA));
    //  CHECK(hipFree(dB));
    //  CHECK(hipFree(dC));
    //  CHECK(hipFree(dANnzPerRow));
    //  CHECK(hipFree(dCsrValA));
    //  CHECK(hipFree(dCsrRowPtrA));
    //  CHECK(hipFree(dCsrColIndA));
 
    //  CHECK_CUSPARSE(hipsparseDestroyMatDescr(Adescr));
    //  CHECK_CUSPARSE(hipsparseDestroy(handle));
 
     return 0;
}