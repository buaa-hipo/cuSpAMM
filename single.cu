#include "hip/hip_runtime.h"

#include "main.h"
float Norm;

//Mutiple-GPU Plan Structure
typedef struct
{
    mytype *h_A, *h_B;
    float *A_normmap,*B_normmap;
    float *h_C;
    hipStream_t stream;
} TGPUplan;

#define MATRIXOFFSETCPY(dst,src,size_row,size_col,off_row,off_col,total_col) \
for(int i=0;i<size_row;i++){ \
    for(int j=0;j<size_col;j++){ \
        dst[i][j]=GETELEMENT21(src,off_row+i,off_col+j,total_col); \
    } \
}

__global__ void unroll_get_Fnorm_pri(const float* __restrict__ A,float *A_normmap,int m,int n,int blockRowOff){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int kId = blockIdx.x;//kernel
    int thId = threadIdx.x;
    __shared__ float sdata[LoNum*LoNum/8/32];

    int valid=0;
    const int myBlockRow = kId / (n/LoNum)+blockRowOff;
    const int myBlockCol = kId % (n/LoNum);
    const int myBlockId = myBlockRow*(n/LoNum)+myBlockCol;
    const int myThreadRow = thId / (LoNum/8);
    const int myThreadCol = thId % (LoNum/8);
    const int myFinalRow = myBlockRow*LoNum+myThreadRow;
    const int myFinalCol = myBlockCol*LoNum+myThreadCol*8;

    //每个线程取1个
    float val;
    valid = id > m*n? 0:1;
    if(valid){
        int tadd = myFinalRow*n+myFinalCol;
        float t1 = A[tadd];
        float t2 = A[tadd+1];
        float t3 = A[tadd+2];
        float t4 = A[tadd+3];
        float t5 = A[tadd+4];
        float t6 = A[tadd+5];
        float t7 = A[tadd+6];
        float t8 = A[tadd+7];
        val = t1*t1+t2*t2+t3*t3+t4*t4+t5*t5+t6*t6+t7*t7+t8*t8;
    } 
    
    #define FULL_MASK 0xffffffff
    for (int offset = 16; offset > 0; offset /= 2){
        val += __shfl_down_sync(FULL_MASK, val, offset);
        // if(thId%32==0) printf("thid=%d warpid=%d inwarpid=%d val=%f\n",thId,thId/32,thId%32,val);
    }       
    if(thId%32==0){
        sdata[thId/32]=val;
        // printf("%d %d val=%f dim=%d\n",thId,thId/32,sdata[thId/32],blockDim.x);
    } 
    
    __syncthreads();
    float r=0;
    
    if (thId < blockDim.x/32)
    {
        // printf("thid=%d val=%f sw[thid]=%f\n",thId,val,sdata[thId]);
        val=sdata[thId];
        // printf("%d %f\n",thId,sdata[thId]);
        for (int offset = blockDim.x/32/2; offset > 0; offset /= 2)
            val += __shfl_down_sync(FULL_MASK, val, offset);
    }
    if(thId==0){
        A_normmap[myBlockId] = sqrt(val); //记得开方
        // printf("pri kid=%d val=%f\n",kId,val);
    } 
}

//4*32
__global__ void unroll_get_Fnorm_FP16(const half* __restrict__ A,float *A_normmap,int m,int n,int blockRowOff){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int kId = blockIdx.x;//kernel
    int thId = threadIdx.x;
    int warpId = thId / 32;

    const int myBlockRow = kId / (n/LoNum)+blockRowOff;
    const int myBlockCol = kId % (n/LoNum);
    const int myBlockId = myBlockRow*(n/LoNum)+myBlockCol;
    
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> chalf_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;
    
    __shared__ half sdata_half[32*32];
    __shared__ float sdata_float[4];
    
    //要算32*32的范数和,块坐标(myBlockRow,myBlockCol)，每个warp算16*16
    int warpi=warpId/2;
    int warpj=warpId%2;
    
    //setFragment(A, 1.0);
    // setFragment(C, 0.0);
    // loadMatrix(B, X+offset);//+矩阵元素平方操作
    // MMA(C, A, B, C);
    wmma::fill_fragment(a_frag, 1.0f);
    wmma::fill_fragment(chalf_frag, 0.0f);
    wmma::load_matrix_sync(b_frag, GETOFF21(A,myBlockRow*LoNum+warpi*16,myBlockCol*LoNum+warpj*16,n), n);
    for (int i = 0; i < b_frag.num_elements; i++) {
        half t=b_frag.x[i];
        b_frag.x[i] = __float2half(__half2float(t) * __half2float(t));
    }
    wmma::mma_sync(chalf_frag, a_frag, b_frag, chalf_frag);
    // copyFromTo(C, A);
    wmma::store_matrix_sync(GETOFF21(sdata_half,warpi*16,warpj*16,32), chalf_frag, 32,wmma::mem_row_major);
    __syncthreads();
    
    wmma::load_matrix_sync(a_frag, GETOFF21(sdata_half,warpi*16,warpj*16,32), 32);
    
    // for (int i = 0; i < a_frag.num_elements; i++) {
    //     // if(warpId==3) printf("thid=%d %f\n",thId,__half2float(a_frag.x[i]));
    // }

    // // setFrament(B, 1.0);
    // // setFrament(C, 0.0);
    // // MMA(C, A, B, C);
    wmma::fill_fragment(b_frag, 1.0f);
    wmma::fill_fragment(c_frag, 0.0f);
    wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
 
    __syncthreads();
    if(thId%32==0){
        sdata_float[warpId]=c_frag.x[0];
    }
    __syncthreads();
    if(thId==0){
        // for(int i=0;i<4;i++){
        //     printf("kid=%d i=%d %f\n",kId,i,sdata_float[i]);
        // }
        A_normmap[myBlockId]=sqrt(sdata_float[0]+sdata_float[1]+sdata_float[2]+sdata_float[3]);
        // printf("unsqrt, %d %f\n",kId,sdata_float[0]+sdata_float[1]+sdata_float[2]+sdata_float[3]);
    }
}

//每个kernel计算C[LoNum,LoNum]
//32个warp
__global__ void get_C_Threads1Element_Mul(const float* __restrict__ A,const float* __restrict__ A_normmap,const float* __restrict__ B,const float* __restrict__ B_normmap,float* C,const int main_row_offset,float Norm){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int kId = blockIdx.x;//kernel
    int thId = threadIdx.x;
    int REDUCECBL = 1<<(int)(log2(CBLMUN*1.0)+1);
    int warpId = thId / 32;
    int warpi = thId % 32;
    float norm_mul;
    const int first16 = 1-warpId/16;

    const int myBlockRow = kId / (N/LoNum) + main_row_offset; 
    const int myBlockCol = kId % (N/LoNum); //负责计算块坐标C[Brow,Bcol]处的块
    const int myBlockRowOff = myBlockRow*LoNum;
    const int myBlockColOff = myBlockCol*LoNum;

    __shared__ int sC_bitmap[CBLMUN*2];
    __shared__ int sC_offset[CBLMUN];
    __shared__ float sA0[LoNum*LoNum],sB0[LoNum*LoNum];
    __shared__ float sA1[LoNum*LoNum],sB1[LoNum*LoNum]; 

    //得出算哪些
    #pragma unroll
    for(int i=thId;i<CBLMUN*2;i+=blockDim.x){
        if(i<(CBLMUN)){
            norm_mul = GETELEMENT21(A_normmap,myBlockRow,i,K/LoNum) * GETELEMENT21(B_normmap,i,myBlockCol,N/LoNum);
            sC_bitmap[i] = norm_mul>Norm? 1:0; //!范数计算有E的浮动误差，应该是6位有效数字
        }
        else{
            sC_bitmap[i] = 0;
        }
    }
    __syncthreads();//不能和下面合并！因为有的线程的b可能没算完就结束了，但是非常费时间

    for(int i=thId;i<CBLMUN;i+=blockDim.x){
        if(sC_bitmap[i]==1){
            int t=0;
            for(int j=0;j<i;j++){
                if(sC_bitmap[j]==1){
                    t++;
                }
            }
            sC_offset[t]=i;
        }
    }
    __syncthreads();

    //reduce算一共有几个非零值,reduce版本只能处理小规模且为2的幂
    for (unsigned int s = REDUCECBL/2; s > 0; s >>= 1) {
		if (thId < s) {
			sC_bitmap[thId] += sC_bitmap[thId + s];
		}
		__syncthreads();
    }
    const int validNum = sC_bitmap[0]; 

    //32warp 预取，前16个取A，后16个取B
    //每个线程取相邻的两个
    int this_b,next_b;
    if(validNum>0){
        this_b=sC_offset[0];
        //每个thread负责的小块，tempid为16warp中的偏移
        const int tempid=thId-16*32*(warpId/16);
        const int tempi=tempid/16;
        const int tempj=tempid%16*2;
        const float* matrix;
        float *smatrix;
        if(first16){
            smatrix=sA0;
            matrix=&GETELEMENT21(A,myBlockRowOff+tempi,this_b*LoNum+tempj,K);
        }
        else{
            smatrix=sB0;
            matrix=&GETELEMENT21(B,this_b*LoNum+tempi,myBlockColOff+tempj,K);
        }
        smatrix[tempid*2]=*(matrix);
        smatrix[tempid*2+1]=*(matrix+1);
        // printf("%d %d data=%f %f\n",tempid,tempid+1,smatrix[tempid],smatrix[tempid+1]);
    }
    else{
        return;
    }
    
    //进循环
    float * A_this_read=sA0;
    float * B_this_read=sB0;
    float * A_this_write=sA1;
    float * B_this_write=sB1;
    const int tempid=thId-32*16*(warpId/16)-32*8*(warpId/24);
    const int tempi=tempid/8;
    const int tempj=tempid%8*4;
    const float* matrix;
    float *smatrix;
    //16个warp，每个线程计算两个最终结果，算Cblock的[ri,rj]和[ri,rj+1]
    int ri=thId/16;
    int rj=thId%16*2;
    float myCresult1=0.0f,myCresult2=0.0f;

    #pragma unroll 
    for(int i=0;i<validNum;i++){
        __syncthreads(); 
        this_b = sC_offset[i];

        if(first16){
            //前16
            //矩阵小块(LoNum,LoNum)乘 每个线程算C内[thId/L,thId%L]处的最后值
            float* mysA = &GETELEMENT21(A_this_read,ri,0,LoNum);//sA第myTR行，sB第myTC列
            float* mysB1 = &GETELEMENT21(B_this_read,0,rj,LoNum);
            float* mysB2 = &GETELEMENT21(B_this_read,0,rj+1,LoNum);
            
            #pragma unroll
            for(int i=0;i<LoNum;i++){ 
                //算横着的两个虽然B要跨列，但是写回global mem时不用跨列
                myCresult1 += *(mysA+i) * *(mysB1+i*LoNum); 
                myCresult2 += *(mysA+i) * *(mysB2+i*LoNum); 
                // if(thId==0) printf("%f %f %f\n",myCresult1,*(mysA+i),*(mysB1+i*LoNum));
            }
        }
        else{
            //后16warp，共同加载share A(mBR)行第b个块,B(mBC)列第b个块
            if(i<validNum-1){
                next_b = sC_offset[i+1];
                const float* matrix;
                float *smatrix;
                if(warpId<24){
                    smatrix=&A_this_write[tempid*4];
                    matrix=&GETELEMENT21(A,myBlockRowOff+tempi,next_b*LoNum+tempj,K);
                }
                else{
                    smatrix=&B_this_write[tempid*4];
                    matrix=&GETELEMENT21(B,next_b*LoNum+tempi,myBlockColOff+tempj,K);
                }
                *(smatrix)=*(matrix);
                *(smatrix+1)=*(matrix+1);
                *(smatrix+2)=*(matrix+2);
                *(smatrix+3)=*(matrix+3);
            }
        }
        
        if(i%2==0){
            A_this_read=sA1;
            B_this_read=sB1;
            A_this_write=sA0;
            B_this_write=sB0;
        }
        else{
            A_this_read=sA0;
            B_this_read=sB0;
            A_this_write=sA1;
            B_this_write=sB1;
        }
    }

    //前16
    if(first16){
        float* add=&GETELEMENT21(C,myBlockRowOff+ri,myBlockColOff+rj,N);
        *(add)=myCresult1;
        *(add+1)=myCresult2;
        // if(myCresult1!=4096) printf("%f\n",myCresult1);
    }
    

}

//4个warp，计算32*32 (4个warp 4*32个线程)
__global__ void get_C_FP16_B32(const half* __restrict__ A,float* A_normmap,const half* __restrict__ B,float* B_normmap,float* C,const int main_row_offset,float Norm){
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    const int kId = blockIdx.x;
    const int thId = threadIdx.x;
    const int warpId = thId/32;
    int REDUCECBL = 1<<(int)(log2(CBLMUN*1.0)+1);

    __shared__ int sC_bitmap[(CBLMUN/4+1)*4*2]; //四字节对齐
    __shared__ int sC_offset[(CBLMUN/4+1)*4];
    __shared__ half st[LoNum*LoNum];
    __shared__ half sA0[LoNum*LoNum],sB0[LoNum*LoNum]; 
    __shared__ half sA1[LoNum*LoNum],sB1[LoNum*LoNum]; 

    float norm_mul,myCresult=0.0f;
    const int myBlockRow = kId / (N/LoNum) + main_row_offset; 
    const int myBlockCol = kId % (N/LoNum); //负责计算块坐标C[Brow,Bcol]处的块
    const int myBlockRowOff = myBlockRow*LoNum;
    const int myBlockColOff = myBlockCol*LoNum;
    
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a0_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b0_frag;
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a1_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b1_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag; //混精
    wmma::fill_fragment(c_frag, 0.0f);

    
    //需要A_norm第R行，B_norm第C列
    for(int i=thId;i<REDUCECBL;i+=blockDim.x){
        if(i<(CBLMUN)){
            norm_mul = GETELEMENT21(A_normmap,myBlockRow,i,K/LoNum) * GETELEMENT21(B_normmap,i,myBlockCol,N/LoNum);
            sC_bitmap[i] = norm_mul>=Norm? 1:0; //!范数计算有E的浮动误差，应该是6位有效数字
        }
        else{
            sC_bitmap[i] = 0;
        }
    }
    __syncthreads();

    

    // reduce
    for(int i=thId;i<CBLMUN;i+=blockDim.x){
        if(sC_bitmap[i]==1){
            int t=0;
            for(int j=0;j<i;j++){
                if(sC_bitmap[j]==1){
                    t++;
                }
            }
            sC_offset[t]=i;
            
        }
        __syncthreads();
    }
    
    //reduce算一共有几个非零值
    for (unsigned int s = REDUCECBL / 2; s > 0; s >>= 1) {
		if (thId < s) {
			sC_bitmap[thId] += sC_bitmap[thId + s];
		}
		__syncthreads();
    }
    const int validNum = sC_bitmap[0]; 
    
    const int warpi=warpId/2;
    const int warpj=warpId%2;
    const int myFinalRow16 = myBlockRow*2+warpi;
    const int myFinalCol16 = myBlockCol*2+warpj;

    int this_b,next_b;
    half * A_this_read=sA0;
    half * B_this_read=sB0;
    half * A_this_write=sA1;
    half * B_this_write=sB1;
    const int inWarpi = thId % 32 / 4 + warpId*8;
    const int inWarpj = (thId % 32 % 4)*8;
    if(validNum>0){
        this_b=sC_offset[0];
        for(int line=warpId*8;line<(warpId+1)*8;line++){
            GETELEMENT21(A_this_read,line,thId%32,LoNum) = GETELEMENT21(A,myBlockRowOff+line,this_b*LoNum+thId%32,K);
            GETELEMENT21(B_this_read,line,thId%32,LoNum) = GETELEMENT21(B,this_b*LoNum+line,myBlockColOff+thId%32,N);
        }
    }
    __syncthreads();

    //遍历bitmap,每个线程负责一个位置的元素
    for(int i=0;i<validNum;i++){
        __syncthreads();
        this_b = sC_offset[i];
        // norm_mul = GETELEMENT21(A_normmap,myBlockRow,i,K/LoNum) * GETELEMENT21(B_normmap,i,myBlockCol,N/LoNum);
        // int vvmul = norm_mul>=Norm? 1:0;
        if(i+1<validNum){
            next_b = sC_offset[i+1];
            __syncthreads();
            #pragma unroll
            for(int line=warpId*8;line<(warpId+1)*8;line++){
                GETELEMENT21(A_this_write,line,thId%32,LoNum) = GETELEMENT21(A,myBlockRowOff+line,next_b*LoNum+thId%32,K);
                GETELEMENT21(B_this_write,line,thId%32,LoNum) = GETELEMENT21(B,next_b*LoNum+line,myBlockColOff+thId%32,N);
                __syncthreads();
            }
        }
        __syncthreads();
        
        wmma::load_matrix_sync(a0_frag, GETOFF21(A_this_read,warpi*16,0*16,LoNum), LoNum);
        wmma::load_matrix_sync(b0_frag, GETOFF21(B_this_read,0*16,warpj*16,LoNum), LoNum);
        wmma::load_matrix_sync(a1_frag, GETOFF21(A_this_read,warpi*16,1*16,LoNum), LoNum);
        wmma::load_matrix_sync(b1_frag, GETOFF21(B_this_read,1*16,warpj*16,LoNum), LoNum);
        wmma::mma_sync(c_frag, a0_frag, b0_frag, c_frag);
        wmma::mma_sync(c_frag, a1_frag, b1_frag, c_frag);

        if(i%2==0){
            A_this_read=sA1;
            B_this_read=sB1;
            A_this_write=sA0;
            B_this_write=sB0;
        }
        else{
            A_this_read=sA0;
            B_this_read=sB0;
            A_this_write=sA1;
            B_this_write=sB1;
        }
        __syncthreads();
    }

    wmma::store_matrix_sync(GETOFF21(C,myFinalRow16*16,myFinalCol16*16,N), c_frag, N,wmma::mem_row_major);

    
}


int main(int argc, char **argv){
    printf("输入参数: M=%d K=%d N=%d Norm=%f USINGHALF=%d\n",M,K,N,NormINIT,USINGHALF);
    printf("初始化输入矩阵...\n");

    int device_row_offset=K/LoNum/DEVICEDIM;
    //测试part是否太大
    if(K/LoNum/DEVICEDIM/PART<=0){
        printf("PART error! too many parts!\n");
        return;
    }

    TGPUplan      plan[DEVICEDIM];
    for(int i=0;i<DEVICEDIM;i++){
        hipSetDevice(i);
        hipStreamCreate(&plan[i].stream);
    }

    //统一内存h_A,h_B;
    mytype *h_A = (mytype *)malloc(sizeof(mytype)*M*K);
    mytype *h_B = (mytype *)malloc(sizeof(mytype)*K*N);
    
    // //给A,B赋值
    if(CNN||DECAY) {
        #if CNN
        getMatrixFromCSV(h_A,M,K,FILENAMEA);
        getMatrixFromCSV(h_B,K,N,FILENAMEB);
        #endif
        #if DECAY
        getMatrixFromMTX(h_A,M,K,FILENAMEA);
        getMatrixFromMTX(h_B,K,N,FILENAMEB);
        #endif
    }
    if(MATRIXNOR) getNormMatrix(h_A,h_B);
    if(MATRIXEXP){
        getDecayMatrixExp(h_A,1,0.1,M,K);
        getDecayMatrixExp(h_B,1,0.1,K,N);
    }
    if(MATRIXALG){
        getDecayMatrixAlg(h_A,0.1,0.1,M,K);
        getDecayMatrixAlg(h_B,0.1,0.1,K,N);
    }
    // printf("---A---\n");MATRIXSHOW21D(h_A,M,K);

    

    printf("初始化输入矩阵完成\n");
    // printf("para: M=%d K=%d N=%d Norm=%f DEVICE=%d PARTS=%d \nALG=%d EXP=%d CNN=%d DECAY=%d\n",M,K,N,NormINIT,DEVICEDIM,PART,MATRIXALG,MATRIXEXP,CNN,DECAY);
    // printf("TUNINGFLAG=%d ExpectedRate=%f TUNINGTIME=%d TUNINGERROR=%f\n",TUNINGFLAG,ExpectedRate,TUNINGTIME,TUNINGERROR);
    for(int i=0;i<DEVICEDIM;i++){
        //给私有的bitmap和C分配空间，C用UM
        hipSetDevice(i);
        hipMallocManaged((void **)&plan[i].h_A, sizeof(mytype)*M*K);
        hipMallocManaged((void **)&plan[i].h_B, sizeof(mytype)*K*N);
        hipMallocManaged((void **)&plan[i].h_C, sizeof(float)*M*N);
        hipMallocManaged((void **)&plan[i].A_normmap, sizeof(float)*(M/LoNum)*(K/LoNum));
        hipMallocManaged((void **)&plan[i].B_normmap, sizeof(float)*(K/LoNum)*(N/LoNum));

        // //UM指导
        // hipMemPrefetchAsync(plan[i].h_A, sizeof(mytype)*M*K, i); 
        // hipMemPrefetchAsync(plan[i].h_B, sizeof(mytype)*K*N, i);
        // hipMemPrefetchAsync(plan[i].h_C, sizeof(float)*M*N, i);
        // hipMemAdvise(plan[i].h_A, sizeof(mytype)*M*K, hipMemAdviseSetReadMostly, i);
        // hipMemAdvise(plan[i].h_B, sizeof(mytype)*K*N, hipMemAdviseSetReadMostly, i);

        //流
        hipStreamCreate(&plan[i].stream);

        //拷贝数据
        hipMemcpy(plan[i].h_A,h_A,sizeof(mytype)*M*K,hipMemcpyHostToDevice);
        hipMemcpy(plan[i].h_B,h_B,sizeof(mytype)*K*N,hipMemcpyHostToDevice);
    }
    
    //计时部分
    hipEvent_t start, stop;
    float elapsed = 0.0;
    double sum=0.0;
    printf("\n***计时***\n");

    #if SpAMM
    for(int i=0;i<=TESTTIME;i++){
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        

        const int partBlockOffset=M/LoNum/PART; //所有行分P次算
        int C_blocks = M*N/(LoNum*LoNum),C_threads=LoNum*LoNum;
        int A_blocks = M*K/(LoNum*LoNum),B_blocks = (K*N)/(LoNum*LoNum),F_threads = LoNum*LoNum;


        #pragma omp parallel num_threads(DEVICEDIM)
        { 
            int device = omp_get_thread_num();
            hipSetDevice(device);

            //计算全部B范数
            for(int p=0;p<PART;p++){
                #if !USINGHALF
                if(LoNum==32){
                    unroll_get_Fnorm_pri<<<B_blocks/PART,F_threads/8,0,plan[device].stream>>>(plan[device].h_B,plan[device].B_normmap,K,N,p*partBlockOffset);
                }
                else{

                }
                
                #else
                if(LoNum==32){
                    unroll_get_Fnorm_FP16<<<B_blocks/PART,32*4,0,plan[device].stream>>>(plan[device].h_B,plan[device].B_normmap,K,N,p*partBlockOffset);
                }
                else{

                }
                #endif
                // printf("---the normmap of B:---\n");
                // // MATRIXSHOW21D(plan[device].B_normmap,B_blocks,1);

                //计算某几行A范数和C结果
                #if !USINGHALF
                if(LoNum==32){
                    unroll_get_Fnorm_pri<<<A_blocks/DEVICEDIM/PART,F_threads/8,0,plan[device].stream>>>(plan[device].h_A,plan[device].A_normmap,M,K,device*(M/LoNum/DEVICEDIM)+p*(partBlockOffset/DEVICEDIM));
                }
                else{

                }
                
                #else
                if(LoNum==32){
                    unroll_get_Fnorm_FP16<<<A_blocks/DEVICEDIM/PART,32*4,0,plan[device].stream>>>(plan[device].h_A,plan[device].A_normmap,M,K,device*(M/LoNum/DEVICEDIM)+p*(partBlockOffset/DEVICEDIM));
                }
                else{

                }
                #endif
            }

            hipStreamSynchronize(plan[device].stream);

            #if TUNINGFLAG
            Norm = tuneValidRate(plan[device].A_normmap,plan[device].B_normmap,M/DEVICEDIM/PART,N);
            #else
            Norm = NormINIT;
            #endif

            for(int p=0;p<PART;p++){
                #if !USINGHALF
                if(LoNum==32){
                    get_C_Threads1Element_Mul<<<C_blocks/DEVICEDIM/PART,C_threads,0,plan[device].stream>>>(plan[device].h_A,plan[device].A_normmap,plan[device].h_B,plan[device].B_normmap,plan[device].h_C,device*(M/LoNum/DEVICEDIM)+p*(partBlockOffset/DEVICEDIM),Norm);
                }
                else{

                }
                
                #else
                if(LoNum==32){
                    get_C_FP16_B32<<<C_blocks/DEVICEDIM/PART,32*4,0,plan[device].stream>>>(plan[device].h_A,plan[device].A_normmap,plan[device].h_B,plan[device].B_normmap,plan[device].h_C,device*(M/LoNum/DEVICEDIM)+p*(partBlockOffset/DEVICEDIM),Norm);
                }
                else{

                }
                #endif
            }
        }

        // //host同步
        hipDeviceSynchronize();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        elapsed /= 1000.0f;
        if(i>WARMUP) sum += elapsed; 
    }

    
    double spammtime=sum/((TESTTIME-WARMUP));
    printf("SpAMM 平均执行时间=%fs\n",spammtime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    #endif
    
    #if CUBLAS
    double cublastime=run_cublas_time(h_A,h_B);
    printf("cuBLAS平均执行时间=%fs\n",cublastime);
    printf("SpAMM加速比=%f\n",cublastime/spammtime);
    #endif
    
    // //检验结果
    #if SpAMM
    if(CHECK) {
        //整合最终C的结果,C永远是float
        float* result_C;
        hipMallocManaged((void **)&result_C, sizeof(float)*M*N);
        for(int i=0;i<M;i++){
            for(int j=0;j<N;j++){
                result_C[i*N+j]=plan[i/(M/DEVICEDIM)].h_C[i*N+j];
            }
        }
        // MATRIXSHOW21D(result_C,M,N);
        // printf("hahah %f\n",result_C[1024]);
        // check_simple_gpu(h_A,h_B,result_C);
        

        //取0号的normmap验证
        float *h_Amap;
        hipMallocManaged((void **)&h_Amap, sizeof(float)*M*K/LoNum/LoNum);
        const int ndim = M*K/LoNum/LoNum/DEVICEDIM;
        for(int device=0;device<DEVICEDIM;device++){
            for(int i=0;i<ndim;i++){
                h_Amap[i+device*ndim] = plan[device].A_normmap[i+device*ndim];
            }
        }
        countValid(h_Amap,plan[0].B_normmap);
        // printf("A norm");
        // checkNormMap(h_A,h_Amap);//测试范数
        // printf("B norm");
        // checkNormMap(h_B,h_Bmap);//测试范数
        
    }
    #endif

    // printf("---NORM squrt A:---\n"); MATRIXSHOW21D(A_normmap,CBLMUN,CBLMUN);
    // printf("---NORM squrt B:---\n"); MATRIXSHOW21D(B_normmap,CBLMUN,CBLMUN);
    // printf("!!! NORM mul setting = %f!!!\n\n",Norm);
    
    //end
    // hipFree(d_A);
    // hipFree(d_B);
    // hipFree(d_C);
}
