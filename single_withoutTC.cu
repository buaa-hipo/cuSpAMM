#include "hip/hip_runtime.h"

#include "main.h"
float Norm;

//Mutiple-GPU Plan Structure
typedef struct
{
    mytype *h_A, *h_B;
    float *A_normmap,*B_normmap;
    float *h_C;
    hipStream_t stream;
} TGPUplan;

#define MATRIXOFFSETCPY(dst,src,size_row,size_col,off_row,off_col,total_col) \
for(int i=0;i<size_row;i++){ \
    for(int j=0;j<size_col;j++){ \
        dst[i][j]=GETELEMENT21(src,off_row+i,off_col+j,total_col); \
    } \
}

__global__ void unroll_get_Fnorm_pri_FP16(const half* __restrict__ A,float *A_normmap,int m,int n,int blockRowOff){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int kId = blockIdx.x;//kernel
    int thId = threadIdx.x;
    __shared__ float sdata[LoNum*LoNum/8/32];

    int valid=0;
    const int myBlockRow = kId / (n/LoNum)+blockRowOff;
    const int myBlockCol = kId % (n/LoNum);
    const int myBlockId = myBlockRow*(n/LoNum)+myBlockCol;
    const int myThreadRow = thId / (LoNum/8);
    const int myThreadCol = thId % (LoNum/8);
    const int myFinalRow = myBlockRow*LoNum+myThreadRow;
    const int myFinalCol = myBlockCol*LoNum+myThreadCol*8;

    //每个线程取1个
    float val;
    valid = id > m*n? 0:1;
    if(valid){
        int tadd = myFinalRow*n+myFinalCol;
        const half t1 = A[tadd];
        const half t2 = A[tadd+1];
        const half t3 = A[tadd+2];
        const half t4 = A[tadd+3];
        const half t5 = A[tadd+4];
        const half t6 = A[tadd+5];
        const half t7 = A[tadd+6];
        const half t8 = A[tadd+7];
        const half t11 = __hmul(t1,t1);
        const half t21 = __hmul(t2,t2);
        const half t31 = __hmul(t3,t3);
        const half t41 = __hmul(t4,t4);
        const half t51 = __hmul(t5,t5);
        const half t61 = __hmul(t6,t6);
        const half t71 = __hmul(t7,t7);
        const half t81 = __hmul(t8,t8);

        const half t12 = __hadd(t11,t21);
        const half t23 = __hadd(t12,t31);
        const half t34 = __hadd(t23,t41);
        const half t45 = __hadd(t34,t51);
        const half t56 = __hadd(t45,t61);
        const half t67 = __hadd(t56,t71);
        const half t78 = __hadd(t67,t81);
        val = __half2float(t78);
    } 
    
    #define FULL_MASK 0xffffffff
    for (int offset = 16; offset > 0; offset /= 2){
        val += __shfl_down_sync(FULL_MASK, val, offset);
        // if(thId%32==0) printf("thid=%d warpid=%d inwarpid=%d val=%f\n",thId,thId/32,thId%32,val);
    }       
    if(thId%32==0){
        sdata[thId/32]=val;
        // printf("%d %d val=%f dim=%d\n",thId,thId/32,sdata[thId/32],blockDim.x);
    } 
    
    __syncthreads();
    float r=0;
    
    if (thId < blockDim.x/32)
    {
        // printf("thid=%d val=%f sw[thid]=%f\n",thId,val,sdata[thId]);
        val=sdata[thId];
        // printf("%d %f\n",thId,sdata[thId]);
        for (int offset = blockDim.x/32/2; offset > 0; offset /= 2)
            val += __shfl_down_sync(FULL_MASK, val, offset);
    }
    if(thId==0){
        A_normmap[myBlockId] = sqrt(val); //记得开方
        // printf("pri kid=%d val=%f\n",kId,val);
    } 
}


//每个kernel计算C[LoNum,LoNum]
//静态无分配版本，每个线程一个元素进行计算，LoNum*LoNum个线程
__global__ void get_C_Threads1Element_Mul_FP16(const half* __restrict__ A,const float* __restrict__ A_normmap,const half* __restrict__ B,const float* __restrict__ B_normmap,float* C,const int main_row_offset,float Norm){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int kId = blockIdx.x;//kernel
    int thId = threadIdx.x;
    int REDUCECBL = 1<<(int)(log2(CBLMUN*1.0)+1);

    __shared__ int sC_bitmap[CBLMUN*2];//share mem需要初始化！！
    // __shared__ int sC_bitmap_debug[CBLMUN];
    __shared__ int sC_offset[CBLMUN];
    __shared__ half sA0[LoNum*LoNum],sB0[LoNum*LoNum]; //sC可以换成局部变量，但有local的风险
    __shared__ half sA1[LoNum*LoNum],sB1[LoNum*LoNum]; 

    float norm_mul;
    half myCresult=__float2half(0.0f);
    const int myBlockRow = kId / (N/LoNum) + main_row_offset; 
    const int myBlockCol = kId % (N/LoNum); //负责计算块坐标C[Brow,Bcol]处的块
    const int myBlockRowOff = myBlockRow*LoNum;
    const int myBlockColOff = myBlockCol*LoNum;
    const int myThreadRow = thId/LoNum;
    const int myThreadCol = thId%LoNum;
    const int myFinalRow = myBlockRowOff+myThreadRow;
    const int myFinalCol = myBlockColOff+myThreadCol;

    // if(thId==0){
    //     printf("kid=%d br=%d bc=%d\n",kId,myBlockRow,myBlockCol);
    // }
    
    
    // 需要A_norm第R行，B_norm第C列
    #pragma unroll
    for(int i=thId;i<REDUCECBL;i+=blockDim.x){
        if(i<(CBLMUN)){
            norm_mul = GETELEMENT21(A_normmap,myBlockRow,i,K/LoNum) * GETELEMENT21(B_normmap,i,myBlockCol,N/LoNum);
            sC_bitmap[i] = norm_mul>Norm? 1:0; //!范数计算有E的浮动误差，应该是6位有效数字
        }
        else{
            sC_bitmap[i] = 0;
        }
    }
    __syncthreads();//不能和下面合并！因为有的线程的b可能没算完就结束了，但是非常费时间


    for(int i=thId;i<CBLMUN;i+=blockDim.x){
        if(sC_bitmap[i]==1){
            int t=0;
            for(int j=0;j<i;j++){
                if(sC_bitmap[j]==1){
                    t++;
                }
            }
            sC_offset[t]=i;
        }
    }
    __syncthreads();

    // //reduce算一共有几个非零值,reduce版本只能处理小规模且为2的幂
    for (unsigned int s = REDUCECBL/2; s > 0; s >>= 1) {
		if (thId < s) {
			sC_bitmap[thId] += sC_bitmap[thId + s];
		}
		__syncthreads();
    }
    const int validNum = sC_bitmap[0]; //不会conflict，只有同bank不同位置才会发生
    // // if(kId==0&&thId==0) printf("\nsum=%d\n",sC_bitmap[0]);

    
    // if(kId==0&&thId==0) printf("\nsum=%d\n",validNum);

    //遍历bitmap,每个线程负责一个位置的元素
    //先使用sA0的数据
    int this_b,next_b;
    if(validNum>0){
        this_b=sC_offset[0];
        sA0[thId] = GETELEMENT21(A,myFinalRow,this_b*LoNum+myThreadCol,K);//慢
        sB0[thId] = GETELEMENT21(B,this_b*LoNum+myThreadRow,myFinalCol,N);
        // if(kId==1&&thId==0) printf("read %d %d\n",myFinalRow,this_b*LoNum+myThreadCol); 
    }
    half * A_this_read=sA0;
    half * B_this_read=sB0;
    half * A_this_write=sA1;
    half * B_this_write=sB1;
    #pragma unroll 
    for(int i=0;i<validNum;i++){
        __syncthreads(); 
        this_b = sC_offset[i];

        //[计算32*32规模的矩阵乘]
        //共同加载share A(mBR)行第b个块,B(mBC)列第b个块
        if(i<validNum-1){
            next_b = sC_offset[i+1];
            A_this_write[thId] = GETELEMENT21(A,myFinalRow,next_b*LoNum+myThreadCol,K);//慢
            B_this_write[thId] = GETELEMENT21(B,next_b*LoNum+myThreadRow,myFinalCol,N);
        }
        
        //矩阵小块(LoNum,LoNum)乘 每个线程算C内[thId/L,thId%L]处的最后值
        half* mysA = &GETELEMENT21(A_this_read,myThreadRow,0,LoNum);//sA第myTR行，sB第myTC列
        half* mysB = &GETELEMENT21(B_this_read,0,myThreadCol,LoNum);
        
        #pragma unroll
        for(int i=0;i<LoNum;i++){ //极慢，三倍
            // myCresult += *(mysA+i) * *(mysB+i*LoNum); 
            const __half a = *(mysA+i);
            const __half b = *(mysB+i*LoNum);
            myCresult +=__hmul(a,b);
            // if(myFinalRow==1186&&myFinalCol==1183){
            //     printf("kid=%d thid=%d b=%d %f %f %f\n",kId,thId,this_b,myCresult,A[1186*K+i],B[i*N+1183]);
            // } 
        }

        if(i%2==0){
            A_this_read=sA1;
            B_this_read=sB1;
            A_this_write=sA0;
            B_this_write=sB0;
        }
        else{
            A_this_read=sA0;
            B_this_read=sB0;
            A_this_write=sA1;
            B_this_write=sB1;
        }
    }

    GETELEMENT21(C,myFinalRow,myFinalCol,N) = myCresult;
}



int main(int argc, char **argv){

    int device_row_offset=K/LoNum/DEVICEDIM;
    //测试part是否太大
    if(K/LoNum/DEVICEDIM/PART<=0){
        printf("PART error! too many parts!\n");
        return;
    }

    TGPUplan      plan[DEVICEDIM];
    for(int i=0;i<DEVICEDIM;i++){
        hipSetDevice(i);
        hipStreamCreate(&plan[i].stream);
    }

    //统一内存h_A,h_B;
    mytype *h_A = (mytype *)malloc(sizeof(mytype)*M*K);
    mytype *h_B = (mytype *)malloc(sizeof(mytype)*K*N);
    printf("%d %d %d\n",M,K,N);
    
    // //给A,B赋值
    if(CNN||DECAY) {
        #if CNN
        getMatrixFromCSV(h_A,M,K,FILENAMEA);
        getMatrixFromCSV(h_B,K,N,FILENAMEB);
        #endif
        #if DECAY
        getMatrixFromMTX(h_A,M,K,FILENAMEA);
        getMatrixFromMTX(h_B,K,N,FILENAMEB);
        #endif
    }
    if(MATRIXNOR) getNormMatrix(h_A,h_B);
    if(MATRIXEXP){
        getDecayMatrixExp(h_A,1,0.1,M,K);
        getDecayMatrixExp(h_B,1,0.1,K,N);
    }
    if(MATRIXALG){
        getDecayMatrixAlg(h_A,0.1,0.1,M,K);
        getDecayMatrixAlg(h_B,0.1,0.1,K,N);
    }
    // printf("---A---\n");MATRIXSHOW21D(h_A,M,K);

    for(int i=0;i<DEVICEDIM;i++){
        //给私有的bitmap和C分配空间，C用UM
        hipSetDevice(i);
        hipMallocManaged((void **)&plan[i].h_A, sizeof(mytype)*M*K);
        hipMallocManaged((void **)&plan[i].h_B, sizeof(mytype)*K*N);
        hipMallocManaged((void **)&plan[i].h_C, sizeof(float)*M*N);
        hipMallocManaged((void **)&plan[i].A_normmap, sizeof(float)*(M/LoNum)*(K/LoNum));
        hipMallocManaged((void **)&plan[i].B_normmap, sizeof(float)*(K/LoNum)*(N/LoNum));

        //UM指导
        hipMemPrefetchAsync(plan[i].h_A, sizeof(mytype)*M*K, i); 
        hipMemPrefetchAsync(plan[i].h_B, sizeof(mytype)*K*N, i);
        hipMemPrefetchAsync(plan[i].h_C, sizeof(float)*M*N, i);
        hipMemAdvise(plan[i].h_A, sizeof(mytype)*M*K, hipMemAdviseSetReadMostly, i);
        hipMemAdvise(plan[i].h_B, sizeof(mytype)*K*N, hipMemAdviseSetReadMostly, i);

        //流
        hipStreamCreate(&plan[i].stream);

        //拷贝数据
        hipMemcpy(plan[i].h_A,h_A,sizeof(mytype)*M*K,hipMemcpyHostToDevice);
        hipMemcpy(plan[i].h_B,h_B,sizeof(mytype)*K*N,hipMemcpyHostToDevice);
    }

    printf("INIT DONE--------------\n");
    printf("para: M=%d K=%d N=%d Norm=%f DEVICE=%d PARTS=%d \nALG=%d EXP=%d CNN=%d DECAY=%d\n",M,K,N,NormINIT,DEVICEDIM,PART,MATRIXALG,MATRIXEXP,CNN,DECAY);
    printf("TUNINGFLAG=%d ExpectedRate=%f TUNINGTIME=%d TUNINGERROR=%f\n",TUNINGFLAG,ExpectedRate,TUNINGTIME,TUNINGERROR);
    //计时部分
    hipEvent_t start, stop;
    float elapsed = 0.0;
    double sum=0.0;

    #if SpAMM
    for(int i=0;i<TESTTIME;i++){
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        #pragma unroll 2
        for(int device=0;device<DEVICEDIM;device++){
            hipSetDevice(device);

            const int partBlockOffset=M/LoNum/PART; //所有行分P次算
            //计算全部B范数
            int A_blocks = M*K/(LoNum*LoNum),B_blocks = (K*N)/(LoNum*LoNum),F_threads = LoNum*LoNum;
            for(int p=0;p<PART;p++){
                #if !USINGHALF
                if(LoNum==32){
                    unroll_get_Fnorm_pri<<<B_blocks/PART,F_threads/8,0,plan[device].stream>>>(plan[device].h_B,plan[device].B_normmap,K,N,p*partBlockOffset);
                }
                else{

                }
                
                #else
                if(LoNum==32){
                    unroll_get_Fnorm_pri_FP16<<<B_blocks/PART,F_threads/8,0,plan[device].stream>>>(plan[device].h_B,plan[device].B_normmap,K,N,p*partBlockOffset);
                }
                else{

                }
                #endif
            }
            hipStreamSynchronize(plan[device].stream);
            // printf("---the normmap of B:---\n");
            // // MATRIXSHOW21D(plan[device].B_normmap,B_blocks,1);


            //计算某几行A范数和C结果
            int C_blocks = M*N/(LoNum*LoNum),C_threads=LoNum*LoNum;
            for(int p=0;p<PART;p++){
                #if !USINGHALF
                if(LoNum==32){
                    unroll_get_Fnorm_pri<<<A_blocks/DEVICEDIM/PART,F_threads/8,0,plan[device].stream>>>(plan[device].h_A,plan[device].A_normmap,M,K,device*(M/LoNum/DEVICEDIM)+p*(partBlockOffset/DEVICEDIM));
                }
                else{

                }
                
                #else
                if(LoNum==32){
                    unroll_get_Fnorm_pri_FP16<<<A_blocks/DEVICEDIM/PART,32*4,0,plan[device].stream>>>(plan[device].h_A,plan[device].A_normmap,M,K,device*(M/LoNum/DEVICEDIM)+p*(partBlockOffset/DEVICEDIM));
                }
                else{

                }
                #endif

                hipStreamSynchronize(plan[device].stream);

                #if TUNINGFLAG
                Norm = tuneValidRate(plan[device].A_normmap,plan[device].B_normmap,M/DEVICEDIM/PART,N);
                #else
                Norm = NormINIT;
                #endif

                #if !USINGHALF
                if(LoNum==32){
                    get_C_Threads1Element_Mul<<<C_blocks/DEVICEDIM/PART,C_threads,0,plan[device].stream>>>(plan[device].h_A,plan[device].A_normmap,plan[device].h_B,plan[device].B_normmap,plan[device].h_C,device*(M/LoNum/DEVICEDIM)+p*(partBlockOffset/DEVICEDIM),Norm);
                }
                else{

                }
                
                #else
                if(LoNum==32){
                    get_C_Threads1Element_Mul_FP16<<<C_blocks/DEVICEDIM/PART,C_threads,0,plan[device].stream>>>(plan[device].h_A,plan[device].A_normmap,plan[device].h_B,plan[device].B_normmap,plan[device].h_C,device*(M/LoNum/DEVICEDIM)+p*(partBlockOffset/DEVICEDIM),Norm);
                }
                else{

                }
                #endif
            }
        }

        // //host同步
        hipDeviceSynchronize();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        elapsed /= 1000.0f;
        if(i>=WARMUP) sum += elapsed; //测速的时候改成3
    }

    printf("spammm time=%fs\n",sum/(TESTTIME-WARMUP));
    hipEventDestroy(start);
    hipEventDestroy(stop);
    #endif
    
    #if CUBLAS
    run_cublas_time(h_A,h_B);
    #endif
    
    // //检验结果
    #if SpAMM
    if(CHECK) {
        //整合最终C的结果,C永远是float
        float* result_C;
        hipMallocManaged((void **)&result_C, sizeof(float)*M*N);
        for(int i=0;i<M;i++){
            for(int j=0;j<N;j++){
                result_C[i*N+j]=plan[i/(M/DEVICEDIM)].h_C[i*N+j];
            }
        }
        // MATRIXSHOW21D(result_C,M,N);
        // printf("hahah %f\n",result_C[1024]);
        check_simple_gpu(h_A,h_B,result_C);
        

        //取0号的normmap验证
        float *h_Amap;
        hipMallocManaged((void **)&h_Amap, sizeof(float)*M*K/LoNum/LoNum);
        const int ndim = M*K/LoNum/LoNum/DEVICEDIM;
        for(int device=0;device<DEVICEDIM;device++){
            for(int i=0;i<ndim;i++){
                h_Amap[i+device*ndim] = plan[device].A_normmap[i+device*ndim];
            }
        }
        countValid(h_Amap,plan[0].B_normmap);
        // printf("A norm");
        // checkNormMap(h_A,h_Amap);//测试范数
        // printf("B norm");
        // checkNormMap(h_B,h_Bmap);//测试范数
        
    }
    #endif

    // printf("---NORM squrt A:---\n"); MATRIXSHOW21D(A_normmap,CBLMUN,CBLMUN);
    // printf("---NORM squrt B:---\n"); MATRIXSHOW21D(B_normmap,CBLMUN,CBLMUN);
    // printf("!!! NORM mul setting = %f!!!\n\n",Norm);
    
    //end
    // hipFree(d_A);
    // hipFree(d_B);
    // hipFree(d_C);
}
