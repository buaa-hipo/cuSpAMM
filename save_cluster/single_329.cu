#include "hip/hip_runtime.h"
//存档
#include "main.h"



#define MATRIXOFFSETCPY(dst,src,size_row,size_col,off_row,off_col,total_col) \
for(int i=0;i<size_row;i++){ \
    for(int j=0;j<size_col;j++){ \
        dst[i][j]=GETELEMENT21(src,off_row+i,off_col+j,total_col); \
    } \
}



#define SHAREFLAG 1
//输入：矩阵A，规模m，n(不能写宏定义因为要复用)，范数锁norm
//每个kernel算[LoNum * LoNum]大小的矩阵范数,每个线程取得一个元素然后reduce
__global__ void get_Fnorm(float *A,float *A_normmap,int m,int n){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int kId = blockIdx.x;//kernel
    int thId = threadIdx.x;
    __shared__ float sdata[LoNum*LoNum];

    int valid;
    float t;
    int myBlockRow = kId / (CBLMUN);
    int myBlockCol = kId % (CBLMUN);
    int myThreadRow = thId / LoNum;
    int myThreadCol = thId % LoNum;

    //每个线程取一个【待优化，可以取多个】
    valid = id > m*n? 0:1;
    if(valid){
        t = GETELEMENT21(A,myBlockRow*LoNum+myThreadRow,myBlockCol*LoNum+myThreadCol,n);
        sdata[thId] = t*t;
    } 
    __syncthreads();

    //naive版reduce，算完完整的范数
    #pragma unroll
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
    {
        if (thId < s)
        {
            sdata[thId] += sdata[thId + s];
        }
         __syncthreads();        // make sure all adds at one stage are done!
    }

    A_normmap[kId] = sqrt(sdata[0]);
}

//每个线程算8个元素,同行相邻的! 每个kernel算一个[LoNum][LoNum]的范数
//每个kernel LoNum*LoNum/8个线程
//!虽然有bank conflict，但是是最快的
__global__ void unroll_get_Fnorm(float *A,float *A_normmap,int m,int n){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int kId = blockIdx.x;//kernel
    int thId = threadIdx.x;
    __shared__ float sdata[LoNum*LoNum];

    int valid;
    int myBlockRow = kId / (CBLMUN);
    int myBlockCol = kId % (CBLMUN);
    int myThreadRow = thId*8 / LoNum;
    int myThreadCol = thId*8 % LoNum;
    int reduceNum = blockDim.x;

    //每个线程取8个
    valid = id*8 > m*n? 0:1;
    if(valid){
        float t1 = GETELEMENT21(A,myBlockRow*LoNum+myThreadRow,myBlockCol*LoNum+myThreadCol,n);
        float t2 = GETELEMENT21(A,myBlockRow*LoNum+myThreadRow,myBlockCol*LoNum+myThreadCol+1,n);
        float t3 = GETELEMENT21(A,myBlockRow*LoNum+myThreadRow,myBlockCol*LoNum+myThreadCol+2,n);
        float t4 = GETELEMENT21(A,myBlockRow*LoNum+myThreadRow,myBlockCol*LoNum+myThreadCol+3,n);
        float t5 = GETELEMENT21(A,myBlockRow*LoNum+myThreadRow,myBlockCol*LoNum+myThreadCol+4,n);
        float t6 = GETELEMENT21(A,myBlockRow*LoNum+myThreadRow,myBlockCol*LoNum+myThreadCol+5,n);
        float t7 = GETELEMENT21(A,myBlockRow*LoNum+myThreadRow,myBlockCol*LoNum+myThreadCol+6,n);
        float t8 = GETELEMENT21(A,myBlockRow*LoNum+myThreadRow,myBlockCol*LoNum+myThreadCol+7,n);
        sdata[thId] = t1*t1+t2*t2+t3*t3+t4*t4+t5*t5+t6*t6+t7*t7+t8*t8;
    } 
    __syncthreads();

    for (unsigned int s = reduceNum / 2; s > 32; s >>= 1) {
		if (thId < s) {
			sdata[thId] += sdata[thId + s];
		}
		__syncthreads();
    }
    
    if (thId < 32)
    {
        volatile float* sw = sdata;
        sw[thId] += sw[thId + 32];
        sw[thId] += sw[thId + 16];
        sw[thId] += sw[thId + 8];
        sw[thId] += sw[thId + 4];
        sw[thId] += sw[thId + 2];
        sw[thId] += sw[thId + 1];
    }
    A_normmap[kId] = sqrt(sdata[0]);
}

//每个kernel计算C[LoNum,LoNum]
//静态无分配版本，每个线程一个元素进行计算
//B normmap !列索引 待优化! ; 分配乘法任务待优化
__global__ void get_C_Threads1Element(float* A,float* A_normmap,float* B,float* B_normmap,float* C){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int kId = blockIdx.x;//kernel
    int thId = threadIdx.x;
    __shared__ int sC_bitmap[CBLMUN];//share mem需要初始化！！
    __shared__ float sA[LoNum*LoNum],sB[LoNum*LoNum],sC[LoNum*LoNum];

    int valid,valid_num=0;
    float norm_mul,t,myCresult=0.0f;
    int myBlockRow = kId / (CBLMUN); 
    int myBlockCol = kId % (CBLMUN); //负责计算块坐标C[Brow,Bcol]处的块

    sC[thId]=0;
    
    //需要A_norm第R行，B_norm第C列
    #pragma unroll
    for(int i=thId;i<CBLMUN;i+=blockDim.x){
        if(thId<(CBLMUN)){
            norm_mul = GETELEMENT21(A_normmap,myBlockRow,i,CBLMUN) * GETELEMENT21(B_normmap,i,myBlockCol,CBLMUN);
            sC_bitmap[i] = norm_mul>Norm? 1:0; //!范数计算有E的浮动误差，应该是6位有效数字
        }
    }
    __syncthreads();//不能和下面合并！因为有的线程的b可能没算完就结束了，但是非常费时间

    //遍历bitmap,每个线程负责一个位置的元素
    #pragma unroll
    for(int b=0;b<CBLMUN;b++){
        if(sC_bitmap[b]==1){
            //共同加载share A(mBR)行第b个块,B(mBC)列第b个块
            sA[thId] = GETELEMENT21(A,myBlockRow*LoNum+thId/LoNum,b*LoNum+thId%LoNum,K);
            sB[thId] = GETELEMENT21(B,b*LoNum+thId/LoNum,myBlockCol*LoNum+thId%LoNum,N);
            __syncthreads();

            //矩阵小块(LoNum,LoNum)乘 每个线程算C内[thId/L,thId%L]处的最后值
            #pragma unroll
            for(int i=0;i<LoNum;i++){
                sC[thId] += GETELEMENT21(sA,thId/LoNum,i,LoNum) * GETELEMENT21(sB,i,thId%LoNum,LoNum);
            }
            __syncthreads();
        }
    }

    //每个线程写回自己负责的块C[Brow,Bcol]里面的值
    //！有conflict 回避不了
    GETELEMENT21(C,myBlockRow*LoNum+thId/LoNum,myBlockCol*LoNum+thId%LoNum,N) = sC[thId]; 
}


int main(int argc, char **argv){
    int deviceId=1;
    // hipGetDevice(&deviceId);
    hipStream_t streams[2];
    if(STREAM){
        for (int i = 0; i < 2; i++) {
            hipStreamCreate(&streams[i]);
        }
    }

    int A_size=M*K*sizeof(float), B_size=K*N*sizeof(float), C_size=M*N*sizeof(float);
    int A_map_num = M*K/(LoNum*LoNum);
    int B_map_num = K*N/(LoNum*LoNum);
    float *h_A,*h_B,*h_C,*d_A,*d_B,*d_C;
    float *A_normmap,*B_normmap;
    float *g_A_normmap,*g_B_normmap;

    if(UNIMEM){
        //统一内存
        hipMallocManaged((void **)&h_A, sizeof(float)*M*K);
        hipMallocManaged((void **)&h_B, sizeof(float)*K*N);
        hipMallocManaged((void **)&h_C, sizeof(float)*M*N);
        hipMalloc((void **)&A_normmap, sizeof(float)*A_map_num);
        hipMalloc((void **)&B_normmap, sizeof(float)*B_map_num); 
        d_A = h_A;
        d_B = h_B;
        d_C = h_C;
    }
    if(PINMEM){
        //锁内存
        hipHostAlloc((void **)&h_A, A_size, hipHostMallocDefault);
        hipHostAlloc((void **)&h_B, B_size, hipHostMallocDefault);
        hipHostAlloc((void **)&h_C, C_size, hipHostMallocDefault);
        hipMalloc((void **)&d_A, A_size);
        hipMalloc((void **)&d_B, B_size);
        hipMalloc((void **)&d_C, C_size);
        hipMalloc((void **)&A_normmap, sizeof(float)*A_map_num);
        hipMalloc((void **)&B_normmap, sizeof(float)*B_map_num);
    }
    // printf("---NORM squrt A:---\n"); MATRIXSHOW21D(A_normmap,1,1);

    if(MATRIXNOR) getNormMatrix(h_A,h_B);
    if(MATRIXEXP){
        getDecayMatrixExp(h_A,1,0.9,M,K);
        getDecayMatrixExp(h_B,1,0.9,K,N);
    }
    if(MATRIXALG){
        getDecayMatrixAlg(h_A,1,0.1,K,N);
        getDecayMatrixAlg(h_B,1,0.1,K,N);
    }
    // printf("---A---\n");MATRIXSHOW21D(h_A,M,K);
    
    //预取
    if(UNIMEM){
        hipMemPrefetchAsync(d_A, A_size, deviceId);
        hipMemPrefetchAsync(d_B, B_size, deviceId);
        hipMemPrefetchAsync(d_C, C_size, deviceId);
        hipMemAdvise(d_A, A_size, hipMemAdviseSetReadMostly, deviceId);
        hipMemAdvise(d_B, B_size, hipMemAdviseSetReadMostly, deviceId);
    }

    //计时部分
    hipEvent_t start, stop;
    float elapsed = 0.0;
    double sum=0.0;

    for(int i=0;i<TESTTIME;i++){
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        if(PINMEM){
            //拷贝数据
            hipMemcpy(d_A, h_A, A_size, hipMemcpyHostToDevice);
            hipMemcpy(d_B, h_B, B_size, hipMemcpyHostToDevice);
        }
        
        //计算范数
        int A_blocks = M*K/(LoNum*LoNum),B_blocks = (K*N)/(LoNum*LoNum),F_threads = LoNum*LoNum;
        if(FORM){
            get_Fnorm<<<A_blocks,F_threads>>>(d_A,A_normmap,M,K);
            get_Fnorm<<<B_blocks,F_threads>>>(d_B,B_normmap,K,N);
        }
        if(UNROLLFORM){
            if(STREAM){
                unroll_get_Fnorm<<<A_blocks,F_threads/8,0,streams[0]>>>(d_A,A_normmap,M,K);
                unroll_get_Fnorm<<<B_blocks,F_threads/8,0,streams[1]>>>(d_B,B_normmap,K,N);
            }
            else{
                unroll_get_Fnorm<<<A_blocks,F_threads/8>>>(d_A,A_normmap,M,K);
                unroll_get_Fnorm<<<B_blocks,F_threads/8>>>(d_B,B_normmap,K,N);
            }
        }
        hipDeviceSynchronize();

        //矩阵乘
        int C_blocks = M*N/(LoNum*LoNum),C_threads=LoNum*LoNum;
        get_C_Threads1Element<<<C_blocks,C_threads>>>(d_A,A_normmap,d_B,B_normmap,d_C);
        hipDeviceSynchronize();
        // printf("---result C:---\n"); MATRIXSHOW21D(h_C,M,N);
        

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        elapsed /= 1000.0f;
        if(i!=0) sum += elapsed;
    }
    
    printf("time=%fs\n",sum/(TESTTIME-1));
    
    hipEventDestroy(start);
    hipEventDestroy(stop);

    //检验结果
    // check_simple_matrix_mul(h_A,h_B,h_C);
    // check(h_A,h_B,h_C);
    float h_Amap[M*K/(LoNum*LoNum)],h_Bmap[K*N/(LoNum*LoNum)];
    hipMemcpy(&h_Amap, A_normmap, sizeof(float)*A_map_num, hipMemcpyDeviceToHost);
    hipMemcpy(&h_Bmap, B_normmap, sizeof(float)*B_map_num, hipMemcpyDeviceToHost);
    countValid(h_Amap,h_Bmap);
    // printf("---NORM squrt A:---\n"); MATRIXSHOW21D(A_normmap,CBLMUN,CBLMUN);
    // printf("---NORM squrt B:---\n"); MATRIXSHOW21D(B_normmap,CBLMUN,CBLMUN);
    // printf("!!! NORM mul setting = %f!!!\n\n",Norm);
    
    //end
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}
