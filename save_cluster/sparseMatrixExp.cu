#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <assert.h>

#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include "main.h"

/*******************/
/* iDivUp FUNCTION */
/*******************/
int iDivUp(int a, int b){ return ((a % b) != 0) ? (a / b + 1) : (a / b); }

/********************/
/* CUDA ERROR CHECK */
/********************/
// --- Credit to http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
void gpuAssert(hipError_t code, const char *file, int line, bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) { exit(code); }
	}
}

void gpuErrchk(hipError_t ans) { gpuAssert((ans), __FILE__, __LINE__); }

/***************************/
/* CUSPARSE ERROR CHECKING */
/***************************/
static const char *_cusparseGetErrorEnum(hipsparseStatus_t error)
{
	switch (error)
	{

	case HIPSPARSE_STATUS_SUCCESS:
		return "HIPSPARSE_STATUS_SUCCESS";

	case HIPSPARSE_STATUS_NOT_INITIALIZED:
		return "HIPSPARSE_STATUS_NOT_INITIALIZED";

	case HIPSPARSE_STATUS_ALLOC_FAILED:
		return "HIPSPARSE_STATUS_ALLOC_FAILED";

	case HIPSPARSE_STATUS_INVALID_VALUE:
		return "HIPSPARSE_STATUS_INVALID_VALUE";

	case HIPSPARSE_STATUS_ARCH_MISMATCH:
		return "HIPSPARSE_STATUS_ARCH_MISMATCH";

	case HIPSPARSE_STATUS_MAPPING_ERROR:
		return "HIPSPARSE_STATUS_MAPPING_ERROR";

	case HIPSPARSE_STATUS_EXECUTION_FAILED:
		return "HIPSPARSE_STATUS_EXECUTION_FAILED";

	case HIPSPARSE_STATUS_INTERNAL_ERROR:
		return "HIPSPARSE_STATUS_INTERNAL_ERROR";

	case HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED:
		return "HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED";

	case HIPSPARSE_STATUS_ZERO_PIVOT:
		return "HIPSPARSE_STATUS_ZERO_PIVOT";
	}

	return "<unknown>";
}

inline void __cusparseSafeCall(hipsparseStatus_t err, const char *file, const int line)
{
	if (HIPSPARSE_STATUS_SUCCESS != err) {
		fprintf(stderr, "CUSPARSE error in file '%s', line %d, error %s\nterminating!\n", __FILE__, __LINE__, \
			_cusparseGetErrorEnum(err)); \
			assert(0); \
	}
}

extern "C" void cusparseSafeCall(hipsparseStatus_t err) { __cusparseSafeCall(err, __FILE__, __LINE__); }

/********/
/* MAIN */
/********/
int main()
{
	hipEvent_t start, stop;
    float elapsed = 0.0;
    float sum=0.0;

	// --- Initialize cuSPARSE
	hipsparseHandle_t handle;	cusparseSafeCall(hipsparseCreate(&handle));

	/**************************/
	/* SETTING UP THE PROBLEM */
	/**************************/
	const int N = T;				// --- Number of rows and columns

	// --- Host side dense matrices
    float *h_A_dense = (float*)malloc(N * N * sizeof(*h_A_dense));
	float *h_B_dense = (float*)malloc(N * N * sizeof(*h_B_dense));
    float *h_C_dense = (float*)malloc(N * N * sizeof(*h_C_dense));
    float *h_A_ori = (float*)malloc(N * N * sizeof(*h_A_dense));
    float *h_B_ori = (float*)malloc(N * N * sizeof(*h_B_dense));
    
    // float* h_A_dense,*h_B_dense,*h_C_dense;
    // float *h_A_ori,*h_B_ori;
    // hipMallocManaged((void **)&h_A_dense, sizeof(float)*M*K);
    // hipMallocManaged((void **)&h_B_dense, sizeof(float)*M*K);
    // hipMallocManaged((void **)&h_C_dense, sizeof(float)*M*K);
    // hipMallocManaged((void **)&h_A_ori, sizeof(float)*M*K);
    // hipMallocManaged((void **)&h_A_ori, sizeof(float)*M*K);
	

	//生成矩阵
	
	if(1){
		printf("alg\n");
        getDecayMatrixAlg(h_A_ori,0.1,0.1,K,N);
        getDecayMatrixAlg(h_B_ori,0.1,0.1,K,N);
    }
    if(0){
		printf("exp\n");
        getDecayMatrixExp(h_A_ori,1,0.9,M,K);
        getDecayMatrixExp(h_B_ori,1,0.9,K,N);
    }
    //截断矩阵
    printf("------------\n");
    printf("para: T=%d truncation=%f\n",T,TRUNCATIONNUM);
	truncation(h_A_dense,h_A_ori,TRUNCATIONNUM);
    truncation(h_B_dense,h_B_ori,TRUNCATIONNUM);

    // MATRIXSHOW21D(h_A_dense,M,N);
    // MATRIXSHOW21D(h_A_ori,M,N);
    

	// --- Create device arrays and copy host arrays to them
	float *d_A_dense;	gpuErrchk(hipMalloc(&d_A_dense, N * N * sizeof(*d_A_dense)));
	float *d_B_dense;	gpuErrchk(hipMalloc(&d_B_dense, N * N * sizeof(*d_B_dense)));
	float *d_C_dense;	gpuErrchk(hipMalloc(&d_C_dense, N * N * sizeof(*d_C_dense)));
	gpuErrchk(hipMemcpy(d_A_dense, h_A_dense, N * N * sizeof(*d_A_dense), hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_B_dense, h_B_dense, N * N * sizeof(*d_B_dense), hipMemcpyHostToDevice));

	// --- Descriptor for sparse matrix A
	hipsparseMatDescr_t descrA;		cusparseSafeCall(hipsparseCreateMatDescr(&descrA));
	cusparseSafeCall(hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL));
	cusparseSafeCall(hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ONE));

	// --- Descriptor for sparse matrix B
	hipsparseMatDescr_t descrB;		cusparseSafeCall(hipsparseCreateMatDescr(&descrB));
	cusparseSafeCall(hipsparseSetMatType(descrB, HIPSPARSE_MATRIX_TYPE_GENERAL));
	cusparseSafeCall(hipsparseSetMatIndexBase(descrB, HIPSPARSE_INDEX_BASE_ONE));

	// --- Descriptor for sparse matrix C
	hipsparseMatDescr_t descrC;		cusparseSafeCall(hipsparseCreateMatDescr(&descrC));
	cusparseSafeCall(hipsparseSetMatType(descrC, HIPSPARSE_MATRIX_TYPE_GENERAL));
	cusparseSafeCall(hipsparseSetMatIndexBase(descrC, HIPSPARSE_INDEX_BASE_ONE));

	int nnzA = 0;							// --- Number of nonzero elements in dense matrix A
	int nnzB = 0;							// --- Number of nonzero elements in dense matrix B

	const int lda = N;						// --- Leading dimension of dense matrix

	// --- Device side number of nonzero elements per row of matrix A
	int *d_nnzPerVectorA; 	gpuErrchk(hipMalloc(&d_nnzPerVectorA, N * sizeof(*d_nnzPerVectorA)));
	cusparseSafeCall(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_COLUMN, N, N, descrA, d_A_dense, lda, d_nnzPerVectorA, &nnzA));

	// --- Device side number of nonzero elements per row of matrix B
	int *d_nnzPerVectorB; 	gpuErrchk(hipMalloc(&d_nnzPerVectorB, N * sizeof(*d_nnzPerVectorB)));
	cusparseSafeCall(hipsparseSnnz(handle, HIPSPARSE_DIRECTION_COLUMN, N, N, descrB, d_B_dense, lda, d_nnzPerVectorB, &nnzB));

	// --- Host side number of nonzero elements per row of matrix A
	int *h_nnzPerVectorA = (int *)malloc(N * sizeof(*h_nnzPerVectorA));
	gpuErrchk(hipMemcpy(h_nnzPerVectorA, d_nnzPerVectorA, N * sizeof(*h_nnzPerVectorA), hipMemcpyDeviceToHost));

	// --- Host side number of nonzero elements per row of matrix B
	int *h_nnzPerVectorB = (int *)malloc(N * sizeof(*h_nnzPerVectorB));
	// gpuErrchk(hipMemcpy(h_nnzPerVectorB, d_nnzPerVectorB, N * sizeof(*h_nnzPerVectorB), hipMemcpyDeviceToHost));

	// printf("Number of nonzero elements in dense matrix A = %i\n\n", nnzA);
	// for (int i = 0; i < N; ++i) printf("Number of nonzero elements in row %i for matrix = %i \n", i, h_nnzPerVectorA[i]);
	// printf("\n");

	// printf("Number of nonzero elements in dense matrix B = %i\n\n", nnzB);
	// for (int i = 0; i < N; ++i) printf("Number of nonzero elements in row %i for matrix = %i \n", i, h_nnzPerVectorB[i]);
	// printf("\n");



	// --- Device side sparse matrix


	float *d_A;			gpuErrchk(hipMalloc(&d_A, nnzA * sizeof(*d_A)));
	float *d_B;			gpuErrchk(hipMalloc(&d_B, nnzB * sizeof(*d_B)));

	int *d_A_RowIndices;	gpuErrchk(hipMalloc(&d_A_RowIndices, (N + 1) * sizeof(*d_A_RowIndices)));
	int *d_B_RowIndices;	gpuErrchk(hipMalloc(&d_B_RowIndices, (N + 1) * sizeof(*d_B_RowIndices)));
	int *d_C_RowIndices;	gpuErrchk(hipMalloc(&d_C_RowIndices, (N + 1) * sizeof(*d_C_RowIndices)));
	int *d_A_ColIndices;	gpuErrchk(hipMalloc(&d_A_ColIndices, nnzA * sizeof(*d_A_ColIndices)));
	int *d_B_ColIndices;	gpuErrchk(hipMalloc(&d_B_ColIndices, nnzB * sizeof(*d_B_ColIndices)));

	cusparseSafeCall(hipsparseSdense2csr(handle, N, N, descrA, d_A_dense, lda, d_nnzPerVectorA, d_A, d_A_RowIndices, d_A_ColIndices));
	cusparseSafeCall(hipsparseSdense2csr(handle, N, N, descrB, d_B_dense, lda, d_nnzPerVectorB, d_B, d_B_RowIndices, d_B_ColIndices));

	// --- Host side sparse matrices
	float *h_A = (float *)malloc(nnzA * sizeof(*h_A));
	float *h_B = (float *)malloc(nnzB * sizeof(*h_B));
	int *h_A_RowIndices = (int *)malloc((N + 1) * sizeof(*h_A_RowIndices));
	int *h_A_ColIndices = (int *)malloc(nnzA * sizeof(*h_A_ColIndices));
	int *h_B_RowIndices = (int *)malloc((N + 1) * sizeof(*h_B_RowIndices));
	int *h_B_ColIndices = (int *)malloc(nnzB * sizeof(*h_B_ColIndices));
	int *h_C_RowIndices = (int *)malloc((N + 1) * sizeof(*h_C_RowIndices));
	gpuErrchk(hipMemcpy(h_A, d_A, nnzA * sizeof(*h_A), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_A_RowIndices, d_A_RowIndices, (N + 1) * sizeof(*h_A_RowIndices), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_A_ColIndices, d_A_ColIndices, nnzA * sizeof(*h_A_ColIndices), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_B, d_B, nnzB * sizeof(*h_B), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_B_RowIndices, d_B_RowIndices, (N + 1) * sizeof(*h_B_RowIndices), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_B_ColIndices, d_B_ColIndices, nnzB * sizeof(*h_B_ColIndices), hipMemcpyDeviceToHost));

	// printf("\nOriginal matrix A in CSR format\n\n");
	// for (int i = 0; i < nnzA; ++i) printf("A[%i] = %f ", i, h_A[i]); printf("\n");

	// printf("\nOriginal matrix B in CSR format\n\n");
	// for (int i = 0; i < nnzB; ++i) printf("B[%i] = %f ", i, h_B[i]); printf("\n");

	// printf("\n");
	// for (int i = 0; i < (N + 1); ++i) printf("h_A_RowIndices[%i] = %i \n", i, h_A_RowIndices[i]); printf("\n");

	// printf("\n");
	// for (int i = 0; i < (N + 1); ++i) printf("h_B_RowIndices[%i] = %i \n", i, h_B_RowIndices[i]); printf("\n");

	// printf("\n");
	// for (int i = 0; i < nnzA; ++i) printf("h_A_ColIndices[%i] = %i \n", i, h_A_ColIndices[i]);

	// printf("\n");
	// for (int i = 0; i < nnzB; ++i) printf("h_B_ColIndices[%i] = %i \n", i, h_B_ColIndices[i]);

	for(int i=0;i<TESTTIME;i++){
        

	// --- Performing the matrix - matrix multiplication
	int baseC, nnzC = 0;
	// nnzTotalDevHostPtr points to host memory
	int *nnzTotalDevHostPtr = &nnzC;

	cusparseSafeCall(hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST));

	cusparseSafeCall(hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, N, descrA, nnzA,
		d_A_RowIndices, d_A_ColIndices, descrB, nnzB, d_B_RowIndices, d_B_ColIndices, descrC, d_C_RowIndices,
		nnzTotalDevHostPtr));
	if (NULL != nnzTotalDevHostPtr) nnzC = *nnzTotalDevHostPtr;
	else {
		gpuErrchk(hipMemcpy(&nnzC, d_C_RowIndices + N, sizeof(int), hipMemcpyDeviceToHost));
		gpuErrchk(hipMemcpy(&baseC, d_C_RowIndices, sizeof(int), hipMemcpyDeviceToHost));
		nnzC -= baseC;
	}
	int *d_C_ColIndices;	gpuErrchk(hipMalloc(&d_C_ColIndices, nnzC * sizeof(int)));
	float *d_C;			gpuErrchk(hipMalloc(&d_C, nnzC * sizeof(float)));
	float *h_C = (float *)malloc(nnzC * sizeof(*h_C));
	int *h_C_ColIndices = (int *)malloc(nnzC * sizeof(*h_C_ColIndices));

		hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);
	cusparseSafeCall(hipsparseScsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, N, N, N, descrA, nnzA,
		d_A, d_A_RowIndices, d_A_ColIndices, descrB, nnzB, d_B, d_B_RowIndices, d_B_ColIndices, descrC,
		d_C, d_C_RowIndices, d_C_ColIndices));

		hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        elapsed /= 1000.0f;
        if(i>0) sum += elapsed;
	
	cusparseSafeCall(hipsparseScsr2dense(handle, N, N, descrC, d_C, d_C_RowIndices, d_C_ColIndices, d_C_dense, N));



	gpuErrchk(hipMemcpy(h_C, d_C, nnzC * sizeof(*h_C), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_C_RowIndices, d_C_RowIndices, (N + 1) * sizeof(*h_C_RowIndices), hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_C_ColIndices, d_C_ColIndices, nnzC * sizeof(*h_C_ColIndices), hipMemcpyDeviceToHost));

	// printf("\nResult matrix C in CSR format\n\n");
	// for (int i = 0; i < nnzC; ++i) printf("C[%i] = %f ", i, h_C[i]); printf("\n");

	// printf("\n");
	// for (int i = 0; i < (N + 1); ++i) printf("h_C_RowIndices[%i] = %i \n", i, h_C_RowIndices[i]); printf("\n");

	// printf("\n");
	// for (int i = 0; i < nnzC; ++i) printf("h_C_ColIndices[%i] = %i \n", i, h_C_ColIndices[i]);

	gpuErrchk(hipMemcpy(h_C_dense, d_C_dense, N * N * sizeof(float), hipMemcpyDeviceToHost));

	// for (int j = 0; j < N; j++) {
	// 	for (int i = 0; i < N; i++)
	// 		printf("%f \t", h_C_dense[i * N + j]);
	// 	printf("\n");
	// }
        

        
    }
    
    printf("time=%fs\n",sum/(TESTTIME-1));
    
    hipEventDestroy(start);
    hipEventDestroy(stop);

	if(CHECK){
        
        check_simple_gpu(h_A_ori,h_B_ori,h_C_dense);
    } 
	
}
