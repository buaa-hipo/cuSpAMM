#include "hip/hip_runtime.h"

#include "main.h"
float Norm;

//Mutiple-GPU Plan Structure
typedef struct
{
    mytype *h_A, *h_B;
    float *A_normmap,*B_normmap;
    float *h_C;
    hipStream_t stream;
} TGPUplan;

#define MATRIXOFFSETCPY(dst,src,size_row,size_col,off_row,off_col,total_col) \
for(int i=0;i<size_row;i++){ \
    for(int j=0;j<size_col;j++){ \
        dst[i][j]=GETELEMENT21(src,off_row+i,off_col+j,total_col); \
    } \
}

__global__ void unroll_get_Fnorm_pri(const float* __restrict__ A,float *A_normmap,int m,int n,int blockRowOff){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int kId = blockIdx.x;
    int thId = threadIdx.x;
    __shared__ float sdata[LoNum*LoNum/8/32];

    int valid=0;
    const int myBlockRow = kId / (n/LoNum)+blockRowOff;
    const int myBlockCol = kId % (n/LoNum);
    const int myBlockId = myBlockRow*(n/LoNum)+myBlockCol;
    const int myThreadRow = thId / (LoNum/8);
    const int myThreadCol = thId % (LoNum/8);
    const int myFinalRow = myBlockRow*LoNum+myThreadRow;
    const int myFinalCol = myBlockCol*LoNum+myThreadCol*8;

    float val;
    valid = id > m*n? 0:1;
    if(valid){
        int tadd = myFinalRow*n+myFinalCol;
        float t1 = A[tadd];
        float t2 = A[tadd+1];
        float t3 = A[tadd+2];
        float t4 = A[tadd+3];
        float t5 = A[tadd+4];
        float t6 = A[tadd+5];
        float t7 = A[tadd+6];
        float t8 = A[tadd+7];
        val = t1*t1+t2*t2+t3*t3+t4*t4+t5*t5+t6*t6+t7*t7+t8*t8;
    } 
    
    #define FULL_MASK 0xffffffff
    for (int offset = 16; offset > 0; offset /= 2){
        val += __shfl_down_sync(FULL_MASK, val, offset);
    }       
    if(thId%32==0){
        sdata[thId/32]=val;
    } 
    
    __syncthreads();
    float r=0;
    
    if (thId < blockDim.x/32)
    {
        val=sdata[thId];
        for (int offset = blockDim.x/32/2; offset > 0; offset /= 2)
            val += __shfl_down_sync(FULL_MASK, val, offset);
    }
    if(thId==0){
        A_normmap[myBlockId] = sqrt(val);
    }
}

//4*32
__global__ void unroll_get_Fnorm_FP16(const half* __restrict__ A,float *A_normmap,int m,int n,int blockRowOff){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int kId = blockIdx.x;
    int thId = threadIdx.x;
    int warpId = thId / 32;

    const int myBlockRow = kId / (n/LoNum)+blockRowOff;
    const int myBlockCol = kId % (n/LoNum);
    const int myBlockId = myBlockRow*(n/LoNum)+myBlockCol;
    
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, half> chalf_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;
    
    __shared__ half sdata_half[32*32];
    __shared__ float sdata_float[4];
    
    int warpi=warpId/2;
    int warpj=warpId%2;
    
    wmma::fill_fragment(a_frag, 1.0f);
    wmma::fill_fragment(chalf_frag, 0.0f);
    wmma::load_matrix_sync(b_frag, GETOFF21(A,myBlockRow*LoNum+warpi*16,myBlockCol*LoNum+warpj*16,n), n);
    for (int i = 0; i < b_frag.num_elements; i++) {
        half t=b_frag.x[i];
        b_frag.x[i] = __float2half(__half2float(t) * __half2float(t));
    }
    wmma::mma_sync(chalf_frag, a_frag, b_frag, chalf_frag);
    
    wmma::store_matrix_sync(GETOFF21(sdata_half,warpi*16,warpj*16,32), chalf_frag, 32,wmma::mem_row_major);
    __syncthreads();
    
    wmma::load_matrix_sync(a_frag, GETOFF21(sdata_half,warpi*16,warpj*16,32), 32);
    
    
    wmma::fill_fragment(b_frag, 1.0f);
    wmma::fill_fragment(c_frag, 0.0f);
    wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
 
    __syncthreads();
    if(thId%32==0){
        sdata_float[warpId]=c_frag.x[0];
    }
    __syncthreads();
    if(thId==0){
        A_normmap[myBlockId]=sqrt(sdata_float[0]+sdata_float[1]+sdata_float[2]+sdata_float[3]);
        
    }
}


__global__ void get_C_Threads1Element_Mul(const float* __restrict__ A,const float* __restrict__ A_normmap,const float* __restrict__ B,const float* __restrict__ B_normmap,float* C,const int main_row_offset,float Norm){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    int kId = blockIdx.x;
    int thId = threadIdx.x;
    int REDUCECBL = 1<<(int)(log2(CBLMUN*1.0)+1);
    int warpId = thId / 32;
    int warpi = thId % 32;
    float norm_mul;
    const int first16 = 1-warpId/16;

    const int myBlockRow = kId / (N/LoNum) + main_row_offset; 
    const int myBlockCol = kId % (N/LoNum);
    const int myBlockRowOff = myBlockRow*LoNum;
    const int myBlockColOff = myBlockCol*LoNum;

    __shared__ int sC_bitmap[CBLMUN*2];
    __shared__ int sC_offset[CBLMUN];
    __shared__ float sA0[LoNum*LoNum],sB0[LoNum*LoNum];
    __shared__ float sA1[LoNum*LoNum],sB1[LoNum*LoNum]; 

    #pragma unroll
    for(int i=thId;i<CBLMUN*2;i+=blockDim.x){
        if(i<(CBLMUN)){
            norm_mul = GETELEMENT21(A_normmap,myBlockRow,i,K/LoNum) * GETELEMENT21(B_normmap,i,myBlockCol,N/LoNum);
            sC_bitmap[i] = norm_mul>Norm? 1:0; 
        }
        else{
            sC_bitmap[i] = 0;
        }
    }
    __syncthreads();

    for(int i=thId;i<CBLMUN;i+=blockDim.x){
        if(sC_bitmap[i]==1){
            int t=0;
            for(int j=0;j<i;j++){
                if(sC_bitmap[j]==1){
                    t++;
                }
            }
            sC_offset[t]=i;
        }
    }
    __syncthreads();

    for (unsigned int s = REDUCECBL/2; s > 0; s >>= 1) {
		if (thId < s) {
			sC_bitmap[thId] += sC_bitmap[thId + s];
		}
		__syncthreads();
    }
    const int validNum = sC_bitmap[0]; 

   
    int this_b,next_b;
    if(validNum>0){
        this_b=sC_offset[0];
        const int tempid=thId-16*32*(warpId/16);
        const int tempi=tempid/16;
        const int tempj=tempid%16*2;
        const float* matrix;
        float *smatrix;
        if(first16){
            smatrix=sA0;
            matrix=&GETELEMENT21(A,myBlockRowOff+tempi,this_b*LoNum+tempj,K);
        }
        else{
            smatrix=sB0;
            matrix=&GETELEMENT21(B,this_b*LoNum+tempi,myBlockColOff+tempj,K);
        }
        smatrix[tempid*2]=*(matrix);
        smatrix[tempid*2+1]=*(matrix+1);
    }
    else{
        return;
    }

    float * A_this_read=sA0;
    float * B_this_read=sB0;
    float * A_this_write=sA1;
    float * B_this_write=sB1;
    const int tempid=thId-32*16*(warpId/16)-32*8*(warpId/24);
    const int tempi=tempid/8;
    const int tempj=tempid%8*4;
    const float* matrix;
    float *smatrix;
    int ri=thId/16;
    int rj=thId%16*2;
    float myCresult1=0.0f,myCresult2=0.0f;

    #pragma unroll 
    for(int i=0;i<validNum;i++){
        __syncthreads(); 
        this_b = sC_offset[i];

        if(first16){

            float* mysA = &GETELEMENT21(A_this_read,ri,0,LoNum);
            float* mysB1 = &GETELEMENT21(B_this_read,0,rj,LoNum);
            float* mysB2 = &GETELEMENT21(B_this_read,0,rj+1,LoNum);
            
            #pragma unroll
            for(int i=0;i<LoNum;i++){ 
                myCresult1 += *(mysA+i) * *(mysB1+i*LoNum); 
                myCresult2 += *(mysA+i) * *(mysB2+i*LoNum); 
            }
        }
        else{
            if(i<validNum-1){
                next_b = sC_offset[i+1];
                const float* matrix;
                float *smatrix;
                if(warpId<24){
                    smatrix=&A_this_write[tempid*4];
                    matrix=&GETELEMENT21(A,myBlockRowOff+tempi,next_b*LoNum+tempj,K);
                }
                else{
                    smatrix=&B_this_write[tempid*4];
                    matrix=&GETELEMENT21(B,next_b*LoNum+tempi,myBlockColOff+tempj,K);
                }
                *(smatrix)=*(matrix);
                *(smatrix+1)=*(matrix+1);
                *(smatrix+2)=*(matrix+2);
                *(smatrix+3)=*(matrix+3);
            }
        }
        
        if(i%2==0){
            A_this_read=sA1;
            B_this_read=sB1;
            A_this_write=sA0;
            B_this_write=sB0;
        }
        else{
            A_this_read=sA0;
            B_this_read=sB0;
            A_this_write=sA1;
            B_this_write=sB1;
        }
    }

    if(first16){
        float* add=&GETELEMENT21(C,myBlockRowOff+ri,myBlockColOff+rj,N);
        *(add)=myCresult1;
        *(add+1)=myCresult2;
    }
    

}

__global__ void get_C_FP16_B32(const half* __restrict__ A,const float* __restrict__ A_normmap,const half* __restrict__ B,const float* __restrict__ B_normmap,float* C,const int main_row_offset,float Norm){
    const int id = blockIdx.x * blockDim.x + threadIdx.x;
    const int kId = blockIdx.x;
    const int thId = threadIdx.x;
    const int warpId = thId/32;
    int REDUCECBL = 1<<(int)(log2(CBLMUN*1.0)+1);

    __shared__ int sC_bitmap[(CBLMUN/4+1)*4*2]; 
    __shared__ int sC_offset[(CBLMUN/4+1)*4];
    __shared__ half st[LoNum*LoNum];
    __shared__ half sA0[LoNum*LoNum],sB0[LoNum*LoNum]; 
    __shared__ half sA1[LoNum*LoNum],sB1[LoNum*LoNum]; 

    float norm_mul,myCresult=0.0f;
    const int myBlockRow = kId / (N/LoNum) + main_row_offset; 
    const int myBlockCol = kId % (N/LoNum); 
    const int myBlockRowOff = myBlockRow*LoNum;
    const int myBlockColOff = myBlockCol*LoNum;
    
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a0_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b0_frag;
    wmma::fragment<wmma::matrix_a, 16, 16, 16, half, wmma::row_major> a1_frag;
    wmma::fragment<wmma::matrix_b, 16, 16, 16, half, wmma::row_major> b1_frag;
    wmma::fragment<wmma::accumulator, 16, 16, 16, float> c_frag;
    wmma::fill_fragment(c_frag, 0.0f);
    
    #pragma unroll
    for(int i=thId;i<REDUCECBL;i+=blockDim.x){
        if(i<(CBLMUN)){
            norm_mul = GETELEMENT21(A_normmap,myBlockRow,i,K/LoNum) * GETELEMENT21(B_normmap,i,myBlockCol,N/LoNum);
            sC_bitmap[i] = norm_mul>=Norm? 1:0; 
        }
        else{
            sC_bitmap[i] = 0;
        }
    }
    __syncthreads();

    // reduce
    for(int i=thId;i<CBLMUN;i+=blockDim.x){
        if(sC_bitmap[i]==1){
            int t=0;
            for(int j=0;j<i;j++){
                if(sC_bitmap[j]==1){
                    t++;
                }
            }
            sC_offset[t]=i;
        }
    }
    __syncthreads();
    
    for (unsigned int s = REDUCECBL / 2; s > 0; s >>= 1) {
		if (thId < s) {
			sC_bitmap[thId] += sC_bitmap[thId + s];
		}
		__syncthreads();
    }
    const int validNum = sC_bitmap[0]; 
    
    const int warpi=warpId/2;
    const int warpj=warpId%2;
    const int myFinalRow16 = myBlockRow*2+warpi;
    const int myFinalCol16 = myBlockCol*2+warpj;

    int this_b,next_b;
    half * A_this_read=sA0;
    half * B_this_read=sB0;
    half * A_this_write=sA1;
    half * B_this_write=sB1;
    const int inWarpi = thId % 32 / 4 + warpId*8;
    const int inWarpj = (thId % 32 % 4)*8;
    if(validNum>0){
        this_b=sC_offset[0];
        #pragma unroll
        for(int line=warpId*8;line<(warpId+1)*8;line++){
            GETELEMENT21(A_this_read,line,thId%32,LoNum) = GETELEMENT21(A,myBlockRowOff+line,this_b*LoNum+thId%32,K);
            GETELEMENT21(B_this_read,line,thId%32,LoNum) = GETELEMENT21(B,this_b*LoNum+line,myBlockColOff+thId%32,N);
        }
    }

    #pragma unroll 
    for(int i=0;i<validNum;i++){
        __syncthreads();
        this_b = sC_offset[i];
        if(i+1<validNum){
            next_b = sC_offset[i+1];
            #pragma unroll
            for(int line=warpId*8;line<(warpId+1)*8;line++){
                GETELEMENT21(A_this_write,line,thId%32,LoNum) = GETELEMENT21(A,myBlockRowOff+line,next_b*LoNum+thId%32,K);
                GETELEMENT21(B_this_write,line,thId%32,LoNum) = GETELEMENT21(B,next_b*LoNum+line,myBlockColOff+thId%32,N);
            }
        }
        
        wmma::load_matrix_sync(a0_frag, GETOFF21(A_this_read,warpi*16,0*16,LoNum), LoNum);
        wmma::load_matrix_sync(b0_frag, GETOFF21(B_this_read,0*16,warpj*16,LoNum), LoNum);
        wmma::load_matrix_sync(a1_frag, GETOFF21(A_this_read,warpi*16,1*16,LoNum), LoNum);
        wmma::load_matrix_sync(b1_frag, GETOFF21(B_this_read,1*16,warpj*16,LoNum), LoNum);
        wmma::mma_sync(c_frag, a0_frag, b0_frag, c_frag);
        wmma::mma_sync(c_frag, a1_frag, b1_frag, c_frag);

        if(i%2==0){
            A_this_read=sA1;
            B_this_read=sB1;
            A_this_write=sA0;
            B_this_write=sB0;
        }
        else{
            A_this_read=sA0;
            B_this_read=sB0;
            A_this_write=sA1;
            B_this_write=sB1;
        }
    }

    wmma::store_matrix_sync(GETOFF21(C,myFinalRow16*16,myFinalCol16*16,N), c_frag, N,wmma::mem_row_major);
}


int main(int argc, char **argv){
    printf("input parameters: M=%d K=%d N=%d Norm=%f USINGHALF=%d\n",M,K,N,NormINIT,USINGHALF);

    int device_row_offset=K/LoNum/DEVICEDIM;
    if(K/LoNum/DEVICEDIM/PART<=0){
        printf("PART error! too many parts!\n");
        return;
    }

    TGPUplan      plan[DEVICEDIM];
    for(int i=0;i<DEVICEDIM;i++){
        hipSetDevice(i);
        hipStreamCreate(&plan[i].stream);
    }

    mytype *h_A = (mytype *)malloc(sizeof(mytype)*M*K);
    mytype *h_B = (mytype *)malloc(sizeof(mytype)*K*N);
    
    if(CNN||DECAY) {
        #if CNN
        getMatrixFromCSV(h_A,M,K,FILENAMEA);
        getMatrixFromCSV(h_B,K,N,FILENAMEB);
        #endif
        #if DECAY
        getMatrixFromMTX(h_A,M,K,FILENAMEA);
        getMatrixFromMTX(h_B,K,N,FILENAMEB);
        #endif
    }
    if(MATRIXNOR) getNormMatrix(h_A,h_B);
    if(MATRIXEXP){
        getDecayMatrixExp(h_A,1,0.1,M,K);
        getDecayMatrixExp(h_B,1,0.1,K,N);
    }
    if(MATRIXALG){
        getDecayMatrixAlg(h_A,0.1,0.1,M,K);
        getDecayMatrixAlg(h_B,0.1,0.1,K,N);
    }
    
    for(int i=0;i<DEVICEDIM;i++){
        hipSetDevice(i);
        hipMallocManaged((void **)&plan[i].h_A, sizeof(mytype)*M*K);
        hipMallocManaged((void **)&plan[i].h_B, sizeof(mytype)*K*N);
        hipMallocManaged((void **)&plan[i].h_C, sizeof(float)*M*N);
        hipMallocManaged((void **)&plan[i].A_normmap, sizeof(float)*(M/LoNum)*(K/LoNum));
        hipMallocManaged((void **)&plan[i].B_normmap, sizeof(float)*(K/LoNum)*(N/LoNum));

        hipMemPrefetchAsync(plan[i].h_A, sizeof(mytype)*M*K, i); 
        hipMemPrefetchAsync(plan[i].h_B, sizeof(mytype)*K*N, i);
        hipMemPrefetchAsync(plan[i].h_C, sizeof(float)*M*N, i);
        hipMemAdvise(plan[i].h_A, sizeof(mytype)*M*K, hipMemAdviseSetReadMostly, i);
        hipMemAdvise(plan[i].h_B, sizeof(mytype)*K*N, hipMemAdviseSetReadMostly, i);

        hipStreamCreate(&plan[i].stream);

        hipMemcpy(plan[i].h_A,h_A,sizeof(mytype)*M*K,hipMemcpyHostToDevice);
        hipMemcpy(plan[i].h_B,h_B,sizeof(mytype)*K*N,hipMemcpyHostToDevice);
    }
    
    hipEvent_t start, stop;
    float elapsed = 0.0;
    double sum=0.0;

    #if SpAMM
    for(int i=0;i<TESTTIME;i++){
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start, 0);

        

        const int partBlockOffset=M/LoNum/PART;
        int C_blocks = M*N/(LoNum*LoNum),C_threads=LoNum*LoNum;
        int A_blocks = M*K/(LoNum*LoNum),B_blocks = (K*N)/(LoNum*LoNum),F_threads = LoNum*LoNum;


        #pragma omp parallel num_threads(DEVICEDIM)
        { 
            int device = omp_get_thread_num();
            hipSetDevice(device);

            for(int p=0;p<PART;p++){
                #if !USINGHALF
                if(LoNum==32){
                    unroll_get_Fnorm_pri<<<B_blocks/PART,F_threads/8,0,plan[device].stream>>>(plan[device].h_B,plan[device].B_normmap,K,N,p*partBlockOffset);
                }
                else{

                }
                
                #else
                if(LoNum==32){
                    unroll_get_Fnorm_FP16<<<B_blocks/PART,32*4,0,plan[device].stream>>>(plan[device].h_B,plan[device].B_normmap,K,N,p*partBlockOffset);
                }
                else{

                }
                #endif
                
                #if !USINGHALF
                if(LoNum==32){
                    unroll_get_Fnorm_pri<<<A_blocks/DEVICEDIM/PART,F_threads/8,0,plan[device].stream>>>(plan[device].h_A,plan[device].A_normmap,M,K,device*(M/LoNum/DEVICEDIM)+p*(partBlockOffset/DEVICEDIM));
                }
                else{

                }
                
                #else
                if(LoNum==32){
                    unroll_get_Fnorm_FP16<<<A_blocks/DEVICEDIM/PART,32*4,0,plan[device].stream>>>(plan[device].h_A,plan[device].A_normmap,M,K,device*(M/LoNum/DEVICEDIM)+p*(partBlockOffset/DEVICEDIM));
                }
                else{

                }
                #endif
            }

            hipStreamSynchronize(plan[device].stream);

            #if TUNINGFLAG
            Norm = tuneValidRate(plan[device].A_normmap,plan[device].B_normmap,M/DEVICEDIM/PART,N);
            #else
            Norm = NormINIT;
            #endif

            for(int p=0;p<PART;p++){
                #if !USINGHALF
                if(LoNum==32){
                    get_C_Threads1Element_Mul<<<C_blocks/DEVICEDIM/PART,C_threads,0,plan[device].stream>>>(plan[device].h_A,plan[device].A_normmap,plan[device].h_B,plan[device].B_normmap,plan[device].h_C,device*(M/LoNum/DEVICEDIM)+p*(partBlockOffset/DEVICEDIM),Norm);
                }
                else{

                }
                
                #else
                if(LoNum==32){
                    get_C_FP16_B32<<<C_blocks/DEVICEDIM/PART,32*4,0,plan[device].stream>>>(plan[device].h_A,plan[device].A_normmap,plan[device].h_B,plan[device].B_normmap,plan[device].h_C,device*(M/LoNum/DEVICEDIM)+p*(partBlockOffset/DEVICEDIM),Norm);
                }
                else{

                }
                #endif
            }
        }

        hipDeviceSynchronize();

        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed, start, stop);
        elapsed /= 1000.0f;
        if(i>WARMUP) sum += elapsed; 
    }

    
    double spammtime=sum/((TESTTIME-WARMUP));
    printf("SpAMM time=%fs\n",spammtime);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    #endif
    
    
}
